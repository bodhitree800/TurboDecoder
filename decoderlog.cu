#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include "hashdefined.h"
//Most of the decoding would be done on the device
//Gamma evaluation on the device
__constant__ int deinter[DATASIZE];
__constant__ int alpha_state_0[8], alpha_state_1[8], alpha_encbit_0[8], alpha_encbit_1[8];
__constant__ int beta_state_0[8], beta_state_1[8], beta_encbit_0[8], beta_encbit_1[8];

void just_decoder_gpu(float snr, int iter_num, int blocks, int guard_size, float *outbits_x, float *outbits_y1, float *outbits_y2, int *decisionbits, int trellis_term, int guarding_type)
{
void result(float LC, float *outbits_x, int *decisionbits,float *Lext12, float *Lext21);
void setup_perm_bits();
void transition_array_creator();
__global__ void gpu_decoder(int blocks, int guard_size, float *Lext_in, float *Lext_out, float *outbits_x, float *outbits_y, float *alpha_guard_read, float *beta_guard_read, float *alpha_guard_write, float *beta_guard_write,  float LC, int which_decoder, int trellis_term, int guarding_type);
__global__ void Lext_initialiser(float *Lext12, float *Lext21);
__global__ void guard_initialiser(int blocks, float *alpha_guard_read_1, float *beta_guard_read_1,  float *alpha_guard_read_2,  float *beta_guard_read_2);
__global__ void sync_guard_values(int blocks, float *alpha_guard_read_1, float *alpha_guard_write_1, float *beta_guard_read_1, float *beta_guard_write_1, float *alpha_guard_read_2, float *alpha_guard_write_2, float *beta_guard_read_2, float *beta_guard_write_2 );
__global__ void result_ker(float LC, float *dev_outbits_x, int *dev_decisionbits, float *dev_Lext12, float *dev_Lext21);
extern int inv_permutation_bits[DATASIZE];
float LC;
int iter;
LC = (4.0/3.0)*pow(10,snr/10.0);
//Allocate memory on the device
//and transferring the required values
float *dev_outbits_x,  *dev_outbits_y1, *dev_outbits_y2;
float *dev_Lext12, *dev_Lext21;
int *dev_decisionbits;
float *dev_alpha_guard_read_1, *dev_alpha_guard_read_2, *dev_beta_guard_read_1, *dev_beta_guard_read_2;
float *dev_alpha_guard_write_1, *dev_alpha_guard_write_2, *dev_beta_guard_write_1, *dev_beta_guard_write_2;

//Allocating memory on the device for outbits and transferring data
hipMalloc((void **)&dev_outbits_x,DATASIZE*sizeof(float));
hipMalloc((void **)&dev_outbits_y1,DATASIZE*sizeof(float));
hipMalloc((void **)&dev_outbits_y2,DATASIZE*sizeof(float));
hipMemcpy(dev_outbits_x,outbits_x,DATASIZE*sizeof(float), hipMemcpyHostToDevice);
hipMemcpy(dev_outbits_y1,outbits_y1,DATASIZE*sizeof(float),hipMemcpyHostToDevice);
hipMemcpy(dev_outbits_y2,outbits_y2,DATASIZE*sizeof(float),hipMemcpyHostToDevice);
//Allocating memory on the device for Lext12 and Lext21
hipMalloc((void **)&dev_Lext12, DATASIZE*sizeof(float));
hipMalloc((void **)&dev_Lext21, DATASIZE*sizeof(float));
//Allocating memory for decisionbits to be set at the end of all this
hipMalloc((void**)&dev_decisionbits, DATASIZE*sizeof(int));
//Allocating memory on the device for alpha and beta guards
hipMalloc((void**)&dev_alpha_guard_read_1, blocks*8*sizeof(float));
hipMalloc((void**)&dev_alpha_guard_read_2, blocks*8*sizeof(float));
hipMalloc((void**)&dev_beta_guard_read_1, blocks*8*sizeof(float));
hipMalloc((void**)&dev_beta_guard_read_2, blocks*8*sizeof(float));
hipMalloc((void**)&dev_alpha_guard_write_1, blocks*8*sizeof(float));
hipMalloc((void**)&dev_alpha_guard_write_2, blocks*8*sizeof(float));
hipMalloc((void**)&dev_beta_guard_write_1, blocks*8*sizeof(float));
hipMalloc((void**)&dev_beta_guard_write_2, blocks*8*sizeof(float));
//Finished allocating memory on the device
Lext_initialiser<<<DATASIZE/64,64>>>(dev_Lext12, dev_Lext21);
setup_perm_bits();
transition_array_creator();
guard_initialiser<<<(blocks*8)/64, 64>>>(blocks, dev_alpha_guard_read_1, dev_beta_guard_read_1,  dev_alpha_guard_read_2,  dev_beta_guard_read_2);
//The iterative decoding is done here
//With repeated calls to kernels on the gpu
//cudaPrintfInit();
for(iter =1; iter<= iter_num; iter++)
{	
	gpu_decoder<<<blocks,8>>>(blocks, guard_size, dev_Lext21, dev_Lext12, dev_outbits_x, dev_outbits_y1, dev_alpha_guard_read_1, dev_beta_guard_read_1, dev_alpha_guard_write_1, dev_beta_guard_write_1, LC, 1, trellis_term, guarding_type);
	gpu_decoder<<<blocks,8>>>(blocks, guard_size, dev_Lext12, dev_Lext21, dev_outbits_x, dev_outbits_y2, dev_alpha_guard_read_2, dev_beta_guard_read_2, dev_alpha_guard_write_2, dev_beta_guard_write_2, LC, 2, trellis_term, guarding_type);
	if(guarding_type != 2)
	{
		sync_guard_values<<<(blocks*8)/64, 64>>>(blocks, dev_alpha_guard_read_1, dev_alpha_guard_write_1, dev_beta_guard_read_1, dev_beta_guard_write_1, dev_alpha_guard_read_2, dev_alpha_guard_write_2, dev_beta_guard_read_2, dev_beta_guard_write_2);
	}
	//cudaPrintfDisplay(stdout, true);
}
//cudaPrintfEnd();
result_ker<<<48,128>>>(LC, dev_outbits_x, dev_decisionbits, dev_Lext12, dev_Lext21);
hipMemcpy(decisionbits, dev_decisionbits, DATASIZE*sizeof(int), hipMemcpyDeviceToHost);

/********Lext12 and Lext21 check block**************
FILE *fp;
int pi;
fp = fopen("Lext_device.dat", "w");
for(pi=0; pi<DATASIZE; pi++)
{
	fprintf(fp, "%f\t%f\t%f\t%f\n", LC*outbits_x[pi], Lext12[pi], Lext21[pi], LC*outbits_x[pi] + Lext12[pi] + Lext21[pi]);
}
fclose(fp);
********Lext12 and Lext21 check block**************/


//Freeing the allocated memory on the device
hipFree(dev_Lext12);
hipFree(dev_Lext21);
hipFree(dev_outbits_x);
hipFree(dev_outbits_y1);
hipFree(dev_outbits_y2);
hipFree(dev_alpha_guard_read_1);
hipFree(dev_alpha_guard_read_2);
hipFree(dev_beta_guard_read_1);
hipFree(dev_beta_guard_read_2);
hipFree(dev_alpha_guard_write_1);
hipFree(dev_alpha_guard_write_2);
hipFree(dev_beta_guard_write_1);
hipFree(dev_beta_guard_write_2);
hipFree(dev_decisionbits);
//Done freeing memory on the device
}

__global__ void gpu_decoder(int blocks, int guard_size, float *Lext_in, float *Lext_out, float *outbits_x, float *outbits_y, float *alpha_guard_read, float *beta_guard_read, float *alpha_guard_write, float *beta_guard_write, float LC, int which_decoder, int trellis_term, int guarding_type)
{
__device__ void guard_alpha_gpu(int blocks, int guard_size, float *start_alpha, float *alpha_guard, float *channel_x_before, float *channel_y_before, float *Lext_before, float LC, int guarding_type);
__device__ void guard_beta_gpu(int blocks, int guard_size, float *start_beta, float *beta_guard, float *channel_x_after, float *channel_y_after, float *Lext_after, float LC, int which_decoder, int trellis_term, int guarding_type);
__device__ int inter(int k);
__device__ float maxstar(float a,float b);
int tid = threadIdx.x;
int bid = blockIdx.x;
int i,n, j, k, l, index, n_eq;
float gamma_0, gamma_1, alpha_0, alpha_1;
float gamma_00, gamma_11, gamma_000, gamma_111;
float beta_0, beta_1;
float Lext_0, Lext_1, Lexternal;


int block_size = DATASIZE/blocks;

//int block_start = bid*block_size;
//int block_end = (bid+1)*block_size - 1;

__shared__ float channel_x[MAX_SUB_BLOCK_SIZE+1], channel_y[MAX_SUB_BLOCK_SIZE+1];
__shared__ float alpha[(MAX_SUB_BLOCK_SIZE+1)*8], Lext[MAX_SUB_BLOCK_SIZE+1];  //520+65*3 715 floats
__shared__ float lambda_0[8*8], lambda_1[8*8]; //128 floats
__shared__ float start_alpha[8], start_beta[8], temp_beta[8], beta[8]; //32 floats
__shared__ float channel_x_before[MAX_GUARD_SIZE], channel_x_after[MAX_GUARD_SIZE], channel_y_before[MAX_GUARD_SIZE], channel_y_after[MAX_GUARD_SIZE]; //64 floats
__shared__ float  Lext_before[MAX_GUARD_SIZE], Lext_after[MAX_GUARD_SIZE]; //32 floats
__shared__ float alpha_guard_copy[8], beta_guard_copy[8]; //16 float

__shared__ float gamma_e1[8], gamma_e2[8];



//************Fecthing into shared memory for the purpose of guarding************
if(guarding_type != 2)
{
	if(bid != 0)
	{
		alpha_guard_copy[tid] = alpha_guard_read[(bid-1)*8+tid];
	}
	if(bid != blocks-1)
	{
		beta_guard_copy[tid] = beta_guard_read[(bid+1)*8+tid];
	}
}
if(guarding_type != 1)
{
	if(bid != 0)
	{
		for(i=0; i<MAX_GUARD_SIZE/8; i++)
		{
			channel_y_before[i*8+tid] = outbits_y[bid*block_size - MAX_GUARD_SIZE + i*8+tid];
		}
		if(which_decoder == 1)
		{
			for(i=0; i<MAX_GUARD_SIZE/8; i++)
			{
				channel_x_before[i*8+tid] = outbits_x[bid*block_size - MAX_GUARD_SIZE + i*8+tid];
				Lext_before[i*8+tid] = Lext_in[bid*block_size - MAX_GUARD_SIZE + i*8+tid];
			}
		}
		else if(which_decoder == 2)
		{
			for(i=0; i<MAX_GUARD_SIZE/8; i++)
			{
				channel_x_before[i*8+tid] = outbits_x[deinter[bid*block_size - MAX_GUARD_SIZE + i*8+tid]];
				Lext_before[i*8+tid] = Lext_in[deinter[bid*block_size - MAX_GUARD_SIZE + i*8+tid]];
			}	
		}
	}
	if(bid != blocks-1)
	{
		for(i=0; i<MAX_GUARD_SIZE/8; i++)
		{
			channel_y_after[i*8+tid] = outbits_y[(bid+1)*block_size +1 + i*8+tid];
		}
		if(which_decoder == 1)
		{
			for(i=0; i<MAX_GUARD_SIZE/8; i++)
			{
				channel_x_after[i*8+tid] = outbits_x[(bid+1)*block_size +1 + i*8+tid];
				Lext_after[i*8+tid] = Lext_in[(bid+1)*block_size +1 + i*8+tid];
			}
		}
		else if(which_decoder == 2)
		{
			for(i=0; i<MAX_GUARD_SIZE/8; i++)
			{
				channel_x_after[i*8+tid] = outbits_x[deinter[(bid+1)*block_size +1 + i*8+tid]];
				Lext_after[i*8+tid] = Lext_in[deinter[(bid+1)*block_size +1 + i*8+tid]];
			}	
		}
	}
}
//************Fecthing into shared memory for the purpose of guarding************
guard_alpha_gpu(blocks, guard_size, start_alpha, alpha_guard_copy, channel_x_before, channel_y_before, Lext_before, LC, guarding_type);
guard_beta_gpu(blocks, guard_size, start_beta, beta_guard_copy, channel_x_after, channel_y_after, Lext_after, LC, which_decoder, trellis_term, guarding_type);
alpha[tid]=start_alpha[tid];
beta[tid] = start_beta[tid];

//*******Performing coalesced memory access to fetch channel values into shared memory******
for(i=0; i<block_size/8; i++)
{
	index = bid*block_size+i*8+tid;
	channel_y[i*8+tid] = outbits_y[index];
}
//*******Performing coalesced memory access to fetch channel values into shared memory******

//********Loading Lext_in appropriately, depending on the which_decoder*********************
for(i=0; i<block_size/8; i++)
{
	index = bid*block_size+i*8+tid;
	if(which_decoder == 1)
	{
		channel_x[i*8+tid] = outbits_x[index];	
		Lext[i*8+tid] = Lext_in[index];
	}
	else if(which_decoder == 2)
	{
		channel_x[i*8+tid] = outbits_x[deinter[index]];
		Lext[i*8+tid] = Lext_in[deinter[index]];
	}
}
//********Loading Lext_in appropriately, depending on the which_decoder*********************

//***********Fetching the values corresponding to first bit of the next block****************
if(bid != blocks-1)
{
	index = (bid+1)*block_size;
	channel_y[block_size] = outbits_y[index];
	if(which_decoder == 1)
	{
		channel_x[block_size] = outbits_x[index];
		Lext[block_size] = Lext_in[index];
	}
	else if(which_decoder == 2)
	{
		channel_x[block_size] = outbits_x[deinter[index]];
		Lext[block_size] = Lext_in[deinter[index]];
	}
}
//***********Fetching the values corresponding to first bit of the next block****************

//*************************************Alpha evaluation*******************************


/*for(i=0; i<block_size; i++)
{	
	
	gamma_0 = (0.5f)*(LC*channel_x[i]+Lext[i])*(-1) + (0.5f)*LC*channel_y[i]*alpha_encbit_0[tid];
	alpha_0 = alpha[ i*8 + alpha_state_0[tid] ] + gamma_0;
	gamma_1 = (0.5f)*(LC*channel_x[i]+Lext[i])*(1) + (0.5f)*LC*channel_y[i]*alpha_encbit_1[tid];
	alpha_1 = alpha[ i*8 + alpha_state_1[tid] ] + gamma_1;
	alpha[ (i+1)*8 + tid ] = maxstar(alpha_0, alpha_1);
	//__syncthreads();
}*/

for(k=0; k<block_size/8; k++)
{
	gamma_e1[tid] = (0.5f)*(LC*channel_x[k*8+tid]+Lext[k*8+tid]);
	gamma_e2[tid] = (0.5f)*LC*channel_y[k*8+tid];
	for(j=0; j<8; j++)
	{	
		i = k*8+j;
		gamma_0 = gamma_e1[j]*(-1) + gamma_e2[j]*alpha_encbit_0[tid];
		gamma_1 = gamma_e1[j] + gamma_e2[j]*alpha_encbit_1[tid];
		alpha_0 = alpha[ i*8 + alpha_state_0[tid] ] + gamma_0;
		alpha_1 = alpha[ i*8 + alpha_state_1[tid] ] + gamma_1;
		alpha[ (i+1)*8 + tid ] = maxstar(alpha_0, alpha_1);
	//__syncthreads();
	}
}

//***************************Writing end alpha for use in the next iteration***************************
if(guarding_type == 1)
{
	alpha_guard_write[bid*8+tid] = alpha[block_size*8+tid];
}
else if(guarding_type == 3)
{
	alpha_guard_write[bid*8+tid] = alpha[(block_size-guard_size)*8+tid];
}

//***************************Writing end alpha for use in the next iteration***************************
//********************Alpha print check********************
//cuPrintf("%f\n", alpha[(2)*8+tid]);
//********************Alpha print check********************
//*************************************Alpha evaluation*******************************

//******************************Beta and Lext evaluation*******************************
if( bid != blocks-1)
{
	gamma_000 = (0.5f)*(LC*channel_x[block_size]+Lext[block_size])*(-1) + (0.5f)*LC*channel_y[block_size]*beta_encbit_0[tid];
	gamma_111 = (0.5f)*(LC*channel_x[block_size]+Lext[block_size])*(1) + (0.5f)*LC*channel_y[block_size]*beta_encbit_1[tid];
	for(k=block_size/8; k>0; k--)
	{
		gamma_e1[tid] = (0.5f)*(LC*channel_x[k*8-tid-1]+Lext[k*8-tid-1]);
		gamma_e2[tid] = (0.5f)*LC*channel_y[k*8-tid-1];
		for(j=0; j<=7; j++)
		{
			n = k*8-j-1;
			//**********************************Beta evaluation****************************
			//gamma_000 = (0.5f)*(LC*channel_x[n+1]+Lext[n+1])*(-1) + (0.5f)*LC*channel_y[n+1]*beta_encbit_0[tid];
			//gamma_111 = (0.5f)*(LC*channel_x[n+1]+Lext[n+1])*(1) + (0.5f)*LC*channel_y[n+1]*beta_encbit_1[tid];
			beta_0 = beta[beta_state_0[tid]] + gamma_000;
			beta_1 = beta[beta_state_1[tid]] + gamma_111;
			temp_beta[tid] = maxstar(beta_0, beta_1);
			beta[tid] = temp_beta[tid];			 
				
			//**********************************Lambda evaluation**************************		
			//gamma_000 = (0.5f)*(LC*channel_x[n]+Lext[n])*(-1) + (0.5f)*LC*channel_y[n]*beta_encbit_0[tid];
			//gamma_111 = (0.5f)*(LC*channel_x[n]+Lext[n])*(1) + (0.5f)*LC*channel_y[n]*beta_encbit_1[tid];
			gamma_000 = gamma_e1[j]*(-1) + gamma_e2[j]*beta_encbit_0[tid];
			gamma_111 = gamma_e1[j] + gamma_e2[j]*beta_encbit_1[tid];
			lambda_0[j*8+tid] = gamma_000 + beta[beta_state_0[tid]] + alpha[n*8+tid];
			lambda_1[j*8+tid] = gamma_111 + beta[beta_state_1[tid]] + alpha[n*8+tid];
			//**********************************Lambda evaluation***************************
			
			//***********************Writing beta for initialisation in the next iteration*************************
			if(guarding_type == 3)
			{
				if(n == guard_size)
				{
					beta_guard_write[bid*8+tid] = beta[tid];
				}
			}	
			//***********************Writing beta for initialisation in the next iteration*************************
			//**********************************Beta evaluation****************************
		
			//****************Print lambda check**********************
			//if(n==block_size-1) cuPrintf("%f\t%f\n", lambda_0[j*8+tid], lambda_1[j*8+tid]);
			//****************Print lambda check**********************
			//********************Beta print check********************
			//if(n== 0) cuPrintf("%f\n", beta[tid]);
			//********************Beta print check********************		
		}
	
		//****************************Lexternal evaluation****************************
		
		Lext_0 = lambda_0[tid*8];
		Lext_1 = lambda_1[tid*8];
		//Lext_0 = 0;
		//Lext_1 = 0;
		for(l=1; l<8; l++)
		{
			//index = tid*8+(l+tid)%8;
			//index = tid*8+(l+tid)&7;
			index = tid*8 + l;
			Lext_0 = maxstar(Lext_0, lambda_0[index]);
			Lext_1 = maxstar(Lext_1, lambda_1[index]);
		}
		n_eq = k*8 - tid - 1;
		Lexternal = (Lext_1 - Lext_0) - LC*channel_x[n_eq] - Lext[n_eq];
		
		//****************Full Lext print check**********************
		//if(k==1) cuPrintf("%f\t%f\t%f\t%f\t%f\n", channel_x[k*8 - tid - 1],Lexternal, Lext_0, Lext_1, Lext[k*8 - tid - 1]);
		//if(k==8) cuPrintf("%f\n", Lexternal);
		//****************Full Lext print check**********************
		
		index = bid*block_size + n_eq;
		if(which_decoder == 1)
		{
			Lext_out[index] = Lexternal; //30 cycles
		}
		else if(which_decoder == 2)
		{
			Lext_out[deinter[index]] = Lexternal; //40 cycles
		}
		//****************************Lexternal evaluation****************************
		//__syncthreads();
		
	}
	//***********************Writing beta for initialisation in the next iteration*************************
	if(guarding_type == 1)
	{
		beta_guard_write[bid*8+tid] = beta[tid];
	}
	//***********************Writing beta for initialisation in the next iteration*************************
	//******************************Beta and Lext evaluation*******************************
}
else
{
	//gamma_00 = (0.5f)*(LC*channel_x[block_size-1]+Lext[block_size-1])*(-1) + (0.5f)*LC*channel_y[block_size-1]*beta_encbit_0[tid];
	//gamma_11 = (0.5f)*(LC*channel_x[block_size-1]+Lext[block_size-1])*(1) + (0.5f)*LC*channel_y[block_size-1]*beta_encbit_1[tid];
	for(k=block_size/8; k>0; k--)
	{
		for(j=0; j<=7; j++)
		{
			n = k*8-j-1;
			if(n == block_size-1)
			{
				temp_beta[tid] = EQUAL_GUARD_VALUE;
				beta[tid] = EQUAL_GUARD_VALUE;
			}
			else
			{
				//**********************************Beta evaluation****************************
				gamma_00 = (0.5f)*(LC*channel_x[n+1]+Lext[n+1])*(-1) + (0.5f)*LC*channel_y[n+1]*beta_encbit_0[tid];
				gamma_11 = (0.5f)*(LC*channel_x[n+1]+Lext[n+1])*(1) + (0.5f)*LC*channel_y[n+1]*beta_encbit_1[tid];
				beta_0 = beta[beta_state_0[tid]] + gamma_00;
				beta_1 = beta[beta_state_1[tid]] + gamma_11;
				temp_beta[tid] = maxstar(beta_0, beta_1);
				beta[tid] = temp_beta[tid];			 
			}
						
			//**********************************Lambda evaluation**************************		
			gamma_00 = (0.5f)*(LC*channel_x[n]+Lext[n])*(-1) + (0.5f)*LC*channel_y[n]*beta_encbit_0[tid];
			gamma_11 = (0.5f)*(LC*channel_x[n]+Lext[n])*(1) + (0.5f)*LC*channel_y[n]*beta_encbit_1[tid];
			lambda_0[j*8+tid] = gamma_00 + beta[beta_state_0[tid]] + alpha[n*8+tid];
			lambda_1[j*8+tid] = gamma_11 + beta[beta_state_1[tid]] + alpha[n*8+tid];
			//**********************************Lambda evaluation***************************
			
		
			//***********************Writing beta for initialisation in the next iteration*************************
			if(guarding_type == 3)
			{
				if(n == guard_size)
				{
					beta_guard_write[bid*8+tid] = beta[tid];
				}
			}	
			//***********************Writing beta for initialisation in the next iteration*************************
			//**********************************Beta evaluation****************************
		
			//****************Print lambda check**********************
			//if(n==block_size-1) cuPrintf("%f\t%f\n", lambda_0[j*8+tid], lambda_1[j*8+tid]);
			//****************Print lambda check**********************
			//********************Beta print check********************
			//if(n== 0) cuPrintf("%f\n", beta[tid]);
			//********************Beta print check********************		
		}
	
		//****************************Lexternal evaluation****************************
		
		Lext_0 = lambda_0[tid*8];
		Lext_1 = lambda_1[tid*8];
		//Lext_0 = 0;
		//Lext_1 = 0;
		for(l=1; l<8; l++)
		{
			//index = tid*8+(l+tid)%8;
			//index = tid*8+(l+tid)&7;
			index = tid*8 + l;
			Lext_0 = maxstar(Lext_0, lambda_0[index]);
			Lext_1 = maxstar(Lext_1, lambda_1[index]);
		}
		n_eq = k*8 - tid - 1;
		Lexternal = (Lext_1 - Lext_0) - LC*channel_x[n_eq] - Lext[n_eq];
		
		//****************Full Lext print check**********************
		//if(k==1) cuPrintf("%f\t%f\t%f\t%f\t%f\n", channel_x[k*8 - tid - 1],Lexternal, Lext_0, Lext_1, Lext[k*8 - tid - 1]);
		//if(k==8) cuPrintf("%f\n", Lexternal);
		//****************Full Lext print check**********************
		
		index = bid*block_size + n_eq;
		if(which_decoder == 1)
		{
			Lext_out[index] = Lexternal; //30 cycles
		}
		else if(which_decoder == 2)
		{
			Lext_out[deinter[index]] = Lexternal; //40 cycles
		}
		//****************************Lexternal evaluation****************************
		//__syncthreads();
		
	}
	//***********************Writing beta for initialisation in the next iteration*************************
	if(guarding_type == 1)
	{
		beta_guard_write[bid*8+tid] = beta[tid];
	}
	//***********************Writing beta for initialisation in the next iteration*************************
	//******************************Beta and Lext evaluation*******************************
}
}

__device__ void guard_alpha_gpu(int blocks, int guard_size, float *start_alpha, float *alpha_guard, float *channel_x_before, float *channel_y_before, float *Lext_before, float LC, int guarding_type)
{
__device__ float maxstar(float a,float b);
int tid = threadIdx.x;
int bid = blockIdx.x;
__shared__ float temp_alpha[8];
int k;
float gamma_0, gamma_1, alpha_0, alpha_1;
if( bid != 0)
{
	if(guarding_type == 1)
	{
		start_alpha[tid] = alpha_guard[tid];
	}
	else if(guarding_type == 2 || guarding_type == 3)
	{
		if(guarding_type == 2)
		{
			start_alpha[tid] = EQUAL_GUARD_VALUE;
		}
		else if(guarding_type == 3)
		{
			start_alpha[tid] = alpha_guard[tid];
		}
		for(k=MAX_GUARD_SIZE-guard_size; k<MAX_GUARD_SIZE; k++)
		{
			
			gamma_0 = (0.5f)*(Lext_before[k] + LC*channel_x_before[k])*(-1) + (0.5f)*LC*channel_y_before[k]*alpha_encbit_0[tid];
			gamma_1 = (0.5f)*(Lext_before[k] + LC*channel_x_before[k])*(1) + (0.5f)*LC*channel_y_before[k]*alpha_encbit_1[tid];
			alpha_0 = start_alpha[alpha_state_0[tid] ] + gamma_0;
			alpha_1 = start_alpha[alpha_state_1[tid] ] + gamma_1;
			temp_alpha[tid] = maxstar(alpha_0, alpha_1);
			//__syncthreads();
			start_alpha[tid] = temp_alpha[tid];
			//__syncthreads();
		}
	}
}
else
{
	if(tid == 0)
	{
		start_alpha[0] = 0;
	}
	else
	{
		start_alpha[tid] = MINUS_INFINITY;
	}
}
}


__device__ void guard_beta_gpu(int blocks, int guard_size, float *start_beta, float *beta_guard, float *channel_x_after, float *channel_y_after, float *Lext_after, float LC, int which_decoder, int trellis_term, int guarding_type)
{
__device__ float maxstar(float a,float b);
int tid = threadIdx.x;
int bid = blockIdx.x;
int k;
float gamma_0, gamma_1, beta_0, beta_1;
__shared__ float temp_beta[8];
if(bid != blocks-1)
{
	if(guarding_type == 1)
	{
		start_beta[tid] = beta_guard[tid];
		
	}
	else if(guarding_type == 2 || guarding_type == 3)
	{
		if(guarding_type == 2)
		{
			start_beta[tid] = EQUAL_GUARD_VALUE;
		}
		else if(guarding_type == 3)
		{
			start_beta[tid] = beta_guard[tid];
		}
		for(k=guard_size-1; k>=0; k--)
		{
			gamma_0 = (0.5f)*(Lext_after[k] + LC*channel_x_after[k])*(-1) + (0.5f)*LC*channel_y_after[k]*beta_encbit_0[tid];
			gamma_1 = (0.5f)*(Lext_after[k] + LC*channel_x_after[k])*(1) + (0.5f)*LC*channel_y_after[k]*beta_encbit_1[tid];
			beta_0 = start_beta[beta_state_0[tid]] + gamma_0;
			beta_1 = start_beta[beta_state_1[tid]] + gamma_1;
			temp_beta[tid] = maxstar(beta_0, beta_1);
			//__syncthreads();
			start_beta[tid] = temp_beta[tid];
			//__syncthreads();
		}	
	}	
}
else
{
	if(which_decoder == 1 && trellis_term == 1)
	{
		if(tid == 0)
		{
			start_beta[0] = 0;
		}
		else
		{
			start_beta[tid] = MINUS_INFINITY;
		}
	}
	else
	{
		start_beta[tid] = EQUAL_GUARD_VALUE;
	}	
}
}



void transition_array_creator()
{
	int cpu_alpha_state_0[8] 	= 	{0, 3, 4, 7, 1, 2, 5, 6};
	int cpu_alpha_state_1[8] 	= 	{1, 2, 5, 6, 0, 3, 4, 7};
	int cpu_beta_state_0[8] 	= 	{0, 4, 5, 1, 2, 6, 7, 3};
	int cpu_beta_state_1[8] 	= 	{4, 0, 1, 5, 6, 2, 3, 7};
	int cpu_alpha_encbit_0[8] 	= 	{-1, 1, 1, -1, -1, 1, 1, -1};
	int cpu_alpha_encbit_1[8] 	= 	{1, -1, -1, 1, 1, -1, -1, 1};
	int cpu_beta_encbit_0[8] 	= 	{-1, -1, 1, 1, 1, 1, -1, -1};
	int cpu_beta_encbit_1[8] 	= 	{1, 1, -1, -1, -1, -1, 1, 1};
	hipMemcpyToSymbol(HIP_SYMBOL(alpha_state_0), cpu_alpha_state_0, 8*sizeof(int));
	hipMemcpyToSymbol(HIP_SYMBOL(alpha_state_1), cpu_alpha_state_1, 8*sizeof(int));
	hipMemcpyToSymbol(HIP_SYMBOL(beta_state_0), cpu_beta_state_0, 8*sizeof(int));
	hipMemcpyToSymbol(HIP_SYMBOL(beta_state_1), cpu_beta_state_1, 8*sizeof(int));
	hipMemcpyToSymbol(HIP_SYMBOL(alpha_encbit_0), cpu_alpha_encbit_0, 8*sizeof(int));
	hipMemcpyToSymbol(HIP_SYMBOL(alpha_encbit_1), cpu_alpha_encbit_1, 8*sizeof(int));
	hipMemcpyToSymbol(HIP_SYMBOL(beta_encbit_0), cpu_beta_encbit_0, 8*sizeof(int));
	hipMemcpyToSymbol(HIP_SYMBOL(beta_encbit_1), cpu_beta_encbit_1, 8*sizeof(int));	
}

void result(float LC, float *outbits_x, int *decisionbits, float *Lext12, float *Lext21)
{
float L1uk;
int k;
for(k=0; k<DATASIZE; k++)
{
	L1uk = LC*outbits_x[k]+ Lext12[k] + Lext21[k];
	//L1uk =Lext12[k];
	//L1uk =Lext12[k];
	//L1uk = Lext12[k] + Lext21[k];
	decisionbits[k] = L1uk > 0 ? 1 : 0;	
}
}


void setup_perm_bits()
{
	extern int inv_permutation_bits[DATASIZE];
	hipMemcpyToSymbol(HIP_SYMBOL(deinter), inv_permutation_bits, DATASIZE*sizeof(int));
}



__device__ int inter(int n)
{
	return (((1+6*n)%DATASIZE)*n)%DATASIZE;
}


__device__ float maxstar(float a,float b)
{
	return max(a,b);
	//return max(a,b) + __logf(1+__expf((-1)*abs(a-b)));
}

__global__ void sync_guard_values(int blocks, float *alpha_guard_read_1, float *alpha_guard_write_1, float *beta_guard_read_1, float *beta_guard_write_1, float *alpha_guard_read_2, float *alpha_guard_write_2, float *beta_guard_read_2, float *beta_guard_write_2 )
{
//Copy write values to the read values
int index = threadIdx.x + blockIdx.x*blockDim.x;
if(index < blocks*8)
{
	alpha_guard_read_1[index] = alpha_guard_write_1[index];
	beta_guard_read_1[index] = beta_guard_write_1[index];
	alpha_guard_read_2[index] = alpha_guard_write_2[index];
	beta_guard_read_2[index] = beta_guard_write_2[index];
}
}


__global__ void guard_initialiser(int blocks, float *alpha_guard_read_1, float *beta_guard_read_1,  float *alpha_guard_read_2,  float *beta_guard_read_2)
{
//Copy write values to the read values
int index = threadIdx.x + blockIdx.x*blockDim.x;
if(index < blocks*8)
{
	alpha_guard_read_1[index] = EQUAL_GUARD_VALUE;
	beta_guard_read_1[index] = EQUAL_GUARD_VALUE;
	alpha_guard_read_2[index] = EQUAL_GUARD_VALUE;
	beta_guard_read_2[index] = EQUAL_GUARD_VALUE;
}
}


__global__ void Lext_initialiser(float *Lext12, float *Lext21)
{
int index = threadIdx.x + blockIdx.x*blockDim.x;
if(index < DATASIZE)
{
	Lext12[index] = 0;
	Lext21[index] = 0;
}
}
__global__ void result_ker(float LC, float *dev_outbits_x, int *dev_decisionbits, float *dev_Lext12, float *dev_Lext21)
{
float L1uk;
int index = threadIdx.x + blockIdx.x*blockDim.x;
if(index < DATASIZE)
{
	L1uk = LC*dev_outbits_x[index] + dev_Lext12[index] + dev_Lext21[index];
	dev_decisionbits[index] = L1uk > 0 ? 1 : 0;	
}
}
