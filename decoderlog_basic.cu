#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include "hashdefined.h"

void just_decoder_basic(float snr, int iter_num, float *outbits_x, float *outbits_y1, float *outbits_y2, int *decisionbits, int trellis_term)
{
void result(float LC, float *outbits_x, int *decisionbits, float *Lext12, float *Lext21);
void basic_decoder(float *Lext_in, float *Lext_out, float *outbits_x, float *outbits_y,  float LC, int which_decoder, int trellis_term);
float Lext12[DATASIZE], Lext21[DATASIZE];
extern int inv_permutation_bits[DATASIZE];
float LC;
int iter;
int i;
LC = (4.0f/3.0f)*pow(10,snr/10.0);
for(i=0; i<DATASIZE; i++)
{
	Lext21[i] = 0;
	Lext12[i] = 0;
}
for(iter =1; iter<= iter_num; iter++)
{
	basic_decoder(Lext21, Lext12, outbits_x, outbits_y1, LC, 1, trellis_term);
	basic_decoder(Lext12, Lext21, outbits_x, outbits_y2, LC, 2, trellis_term);
	
}
result(LC, outbits_x, decisionbits, Lext12, Lext21);
}


void basic_decoder(float *Lext_in, float *Lext_out, float *outbits_x, float *outbits_y,  float LC, int which_decoder, int trellis_term)
{
extern int inv_permutation_bits[DATASIZE];
float maxf(float a, float b);
void guard_alpha(float *start_alpha);
void guard_beta(float *start_beta, int which_decoder, int trellis_term);
int cpu_alpha_state_0[8] 	= 	{0, 3, 4, 7, 1, 2, 5, 6};
int cpu_alpha_state_1[8] 	= 	{1, 2, 5, 6, 0, 3, 4, 7};
int cpu_beta_state_0[8] 	= 	{0, 4, 5, 1, 2, 6, 7, 3};
int cpu_beta_state_1[8] 	= 	{4, 0, 1, 5, 6, 2, 3, 7};
int cpu_alpha_encbit_0[8] 	= 	{-1, 1, 1, -1, -1, 1, 1, -1};
int cpu_alpha_encbit_1[8] 	= 	{1, -1, -1, 1, 1, -1, -1, 1};
int cpu_beta_encbit_0[8] 	= 	{-1, -1, 1, 1, 1, 1, -1, -1};
int cpu_beta_encbit_1[8] 	= 	{1, 1, -1, -1, -1, -1, 1, 1};
float alpha[(DATASIZE+1)*8];
float Lext_input[DATASIZE], channel_x[DATASIZE], channel_y[DATASIZE];
float beta[8], temp_beta[8];
float start_alpha[8], start_beta[8];
float gamma_0, gamma_1;
float alpha_0, alpha_1;
float beta_0, beta_1;
float splus, sminus;
float Lexternal;
int i, j;


for(i=0; i<DATASIZE; i++)
{
	channel_y[i] = outbits_y[i];
}
if(which_decoder == 1)
{
	for(i=0; i<DATASIZE; i++)
	{
		Lext_input[i] = Lext_in[i];
		channel_x[i] = outbits_x[i];
	}
}
else if(which_decoder == 2)
{
	for(i=0; i<DATASIZE; i++)
	{
		Lext_input[i] = Lext_in[inv_permutation_bits[i]];
		channel_x[i] = outbits_x[inv_permutation_bits[i]];
	}
}

guard_alpha(start_alpha);
guard_beta(start_beta, which_decoder, trellis_term);

for(j=0; j<8; j++)
{
	alpha[j] = start_alpha[j];
	beta[j] = start_beta[j];
}

//*******************************************Alpha evaluation**********************************************
for(i=0; i<DATASIZE; i++)
{
	for(j=0; j<8; j++)
	{
		gamma_0 = 0.5*(Lext_input[i] + LC*channel_x[i])*(-1) + 0.5*LC*channel_y[i]*cpu_alpha_encbit_0[j];
		gamma_1 = 0.5*(Lext_input[i] + LC*channel_x[i])*(1) + 0.5*LC*channel_y[i]*cpu_alpha_encbit_1[j];
		alpha_0 = alpha[i*8 + cpu_alpha_state_0[j] ] + gamma_0;
		alpha_1 = alpha[i*8 + cpu_alpha_state_1[j] ] + gamma_1;
		alpha[(i+1)*8 + j] = maxf(alpha_0, alpha_1);
	}
}
//*******************************************Alpha evaluation**********************************************
/************Print alpha test block*******************
FILE *fp1, *fp2, *fp3, *fp4;
fp1 = fopen("alpha_check.dat", "w");
fp2 = fopen("beta_check.dat", "w");
fp3 = fopen("Lext_check.dat", "w");
fp4 = fopen("Full_Lext_check.dat", "w");
for(i=0; i<DATASIZE; i++)
{
	for(j=0; j<8; j++)
	{
		fprintf(fp1, "%f\t", alpha[(i+1)*8+j]);
	}
	fprintf(fp1, "\n");
}
fclose(fp1);
************Print alpha test block*******************/


//*******************************************Beta and Lext evaluation**********************************************
for(i=DATASIZE-1; i>=0; i--)
{
	//***************************************Beta evaluation***********************************************
	if(i == DATASIZE-1)
	{
		for(j=0; j<8; j++)
		{
			beta[j] = EQUAL_GUARD_VALUE;
		}
	}
	else
	{
		for(j=0; j<8; j++)
		{
			gamma_0 = 0.5*(Lext_input[i+1] + LC*channel_x[i+1])*(-1) + 0.5*LC*channel_y[i+1]*cpu_beta_encbit_0[j];
			gamma_1 = 0.5*(Lext_input[i+1] + LC*channel_x[i+1])*(1) + 0.5*LC*channel_y[i+1]*cpu_beta_encbit_1[j];
			beta_0 = beta[cpu_beta_state_0[j] ] + gamma_0;
			beta_1 = beta[cpu_beta_state_1[j] ] + gamma_1;
			temp_beta[j] = maxf(beta_0, beta_1);
		}	
		for(j=0; j<8; j++)
		{
			beta[j] = temp_beta[j];
		}
	}
	/********Print beta block**********
	for(j=0; j<8; j++)
	{
		fprintf(fp2,"%f\t", beta[j]);
	}
	fprintf(fp2, "\n");
	********Print beta block**********/
	//***************************************Beta evaluation***********************************************

	//***************************************Lexternal evaluation******************************************	
	sminus = 0;
	splus =  0;
	for(j=0; j<8; j++)
	{
		gamma_0 = 0.5*(Lext_input[i] + LC*channel_x[i])*(-1) + 0.5*LC*channel_y[i]*cpu_beta_encbit_0[j];
		gamma_1 = 0.5*(Lext_input[i] + LC*channel_x[i])*(1) + 0.5*LC*channel_y[i]*cpu_beta_encbit_1[j];
		sminus = maxf(sminus, alpha[i*8+j] + beta[cpu_beta_state_0[j]] + gamma_0);
		splus = maxf(splus, alpha[i*8+j] + beta[cpu_beta_state_1[j]] + gamma_1); 
	}
	Lexternal = splus - sminus - LC*channel_x[i]- Lext_input[i];
	//Lexternal = Lext_input[i];
	//Lexternal = LC*channel_x[i];
	/*************************************************Mega Lext check*****************************************************************
	fprintf(fp4, "%f\t%f\t%f\t%f\t%f\t%f\t%f\n", channel_x[i], Lexternal,sminus, splus,splus - sminus, LC*channel_x[i], Lext_input[i]);
	*************************************************Mega Lext check*****************************************************************/
	//***************************************Lexternal evaluation******************************************
	if(which_decoder == 1)
	{
		Lext_out[i] = Lexternal;
	}
	else if(which_decoder == 2)
	{
		Lext_out[inv_permutation_bits[i]] = Lexternal;
	}
}
//*******************************************Beta and Lext evaluation**********************************************
/************Printing Lexternal*****************
for(i=0; i<DATASIZE; i++)
{
	fprintf(fp3, "%f\n", Lext_out[i]);
}
fclose(fp2);
fclose(fp3);
fclose(fp4);
************Printing Lexternal*****************/
}


void guard_alpha(float *start_alpha)
{
int i;
start_alpha[0] = 0;
for(i=1; i<8; i++)
{	
	start_alpha[i] = MINUS_INFINITY;
}
}

void guard_beta(float *start_beta, int which_decoder, int trellis_term)
{
int i;
if(which_decoder == 1 && trellis_term == 1)
{
	start_beta[0] = 0;
	for(i=1; i<8; i++)
	{
		start_beta[i] = MINUS_INFINITY;
	}
}
else
{
	for(i=0; i<8; i++)
	{
		start_beta[i] = EQUAL_GUARD_VALUE;
	}
}
}

float maxf(float a, float b)
{
float myabs(float a);
float mymax(float a, float b);
return mymax(a,b);
//return mymax(a,b) + log(1+exp((-1)*myabs(a-b)));
}




float myabs(float a)
{
if(a>=0) return a;
else return (-1)*a;
}

float mymax(float a, float b)
{
if(a>=b)
	return a;
else 
	return b;
}


