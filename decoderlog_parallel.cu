#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include "hashdefined.h"

void just_decoder_parallel(float snr, int iter_num, int blocks, int guard_size, float *outbits_x, float *outbits_y1, float *outbits_y2, int *decisionbits, int trellis_term, int guarding_type)
{
void result(float LC, float *outbits_x, int *decisionbits, float *Lext12, float *Lext21);
void parallel_decoder(int blocks, int guard_size, float *Lext_in, float *Lext_out, float *outbits_x, float *outbits_y, float *read_ag, float *read_bg , float *write_ag, float *write_bg, float LC, int which_decoder, int trellis_term, int guarding_type);
float Lext12[DATASIZE], Lext21[DATASIZE];
float *read_alpha_guard_1, *read_beta_guard_1, *read_alpha_guard_2, *read_beta_guard_2;
float *write_alpha_guard_1, *write_beta_guard_1, *write_alpha_guard_2, *write_beta_guard_2;
extern int inv_permutation_bits[DATASIZE];
float LC;
int iter;
int i;

read_alpha_guard_1 = (float*)malloc(blocks*8*sizeof(float));
read_beta_guard_1 = (float*)malloc(blocks*8*sizeof(float));
read_alpha_guard_2 = (float*)malloc(blocks*8*sizeof(float));
read_beta_guard_2 = (float*)malloc(blocks*8*sizeof(float));
write_alpha_guard_1 = (float*)malloc(blocks*8*sizeof(float));
write_beta_guard_1 = (float*)malloc(blocks*8*sizeof(float));
write_alpha_guard_2 = (float*)malloc(blocks*8*sizeof(float));
write_beta_guard_2 = (float*)malloc(blocks*8*sizeof(float));
for(i=0; i<blocks*8; i++)
{
	read_alpha_guard_1[i] = EQUAL_GUARD_VALUE;
	read_alpha_guard_2[i] = EQUAL_GUARD_VALUE;
	read_beta_guard_1[i] = EQUAL_GUARD_VALUE;
	read_beta_guard_2[i] = EQUAL_GUARD_VALUE;
}

LC = (4.0/3.0)*pow(10,snr/10.0);
for(i=0; i<DATASIZE; i++)
{
	Lext21[i] = 0;
	Lext12[i] = 0;
}
for(iter =1; iter<= iter_num; iter++)
{
	parallel_decoder(blocks, guard_size, Lext21, Lext12, outbits_x, outbits_y1, read_alpha_guard_1, read_beta_guard_1, write_alpha_guard_1, write_beta_guard_1, LC, 1, trellis_term, guarding_type);
	parallel_decoder(blocks, guard_size, Lext12, Lext21, outbits_x, outbits_y2, read_alpha_guard_2, read_beta_guard_2, write_alpha_guard_2, write_beta_guard_2, LC, 2, trellis_term, guarding_type);
}
result(LC, outbits_x, decisionbits, Lext12, Lext21);
}


void parallel_decoder(int blocks, int guard_size, float *Lext_in, float *Lext_out, float *outbits_x, float *outbits_y, float *read_ag, float *read_bg , float *write_ag, float *write_bg, float LC, int which_decoder, int trellis_term, int guarding_type)
{
void figment_decoder(int figment, int blocks, int guard_size, float *Lext_in, float *Lext_out, float *channel_x, float *channel_y, float *read_ag, float *read_bg, float *write_ag, float *write_bg, float LC, int which_decoder, int trellis_term, int guarding_type);
int i;
//******************************************Using the figment decoder to do parallel decoding******************************************
for(i=0; i<blocks; i++)
{
	figment_decoder(i, blocks, guard_size, Lext_in, Lext_out, outbits_x, outbits_y, read_ag, read_bg, write_ag, write_bg, LC, which_decoder, trellis_term, guarding_type);
}
//Synchronizing read, write : alphas and betas
for(i=0; i<blocks*8; i++)
{
	read_ag[i] = write_ag[i];
	read_bg[i] = write_bg[i];
}
//Synchronizing read, write : alphas and betas
//******************************************Using the figment decoder to do parallel decoding******************************************
}

void figment_decoder(int figment, int blocks, int guard_size, float *Lext_in, float *Lext_out, float *outbits_x, float *outbits_y, float *read_ag, float *read_bg, float *write_ag, float *write_bg, float LC, int which_decoder, int trellis_term, int guarding_type)
{
extern int inv_permutation_bits[DATASIZE];
float maxf(float a, float b);
void p_guard_alpha(float *start_alpha, float *guard_alpha, int figment, int guard_size, float *channel_x_before, float *channel_y_before, float *Lext_before, float LC, int guarding_type);
void p_guard_beta(float *start_beta, float *guard_beta, int figment, int guard_size, float *channel_x_after, float *channel_y_after, float *Lext_after, float LC, int blocks, int which_decoder, int trellis_term, int guarding_type);
int cpu_alpha_state_0[8] 	= 	{0, 3, 4, 7, 1, 2, 5, 6};
int cpu_alpha_state_1[8] 	= 	{1, 2, 5, 6, 0, 3, 4, 7};
int cpu_beta_state_0[8] 	= 	{0, 4, 5, 1, 2, 6, 7, 3};
int cpu_beta_state_1[8] 	= 	{4, 0, 1, 5, 6, 2, 3, 7};
int cpu_alpha_encbit_0[8] 	= 	{-1, 1, 1, -1, -1, 1, 1, -1};
int cpu_alpha_encbit_1[8] 	= 	{1, -1, -1, 1, 1, -1, -1, 1};
int cpu_beta_encbit_0[8] 	= 	{-1, -1, 1, 1, 1, 1, -1, -1};
int cpu_beta_encbit_1[8] 	= 	{1, 1, -1, -1, -1, -1, 1, 1};
int block_size;
float *alpha, *Lext_input, *channel_x, *channel_y;
float *Lext_before, *Lext_after, *channel_x_before, *channel_x_after, *channel_y_before, *channel_y_after;
block_size = DATASIZE/blocks;
alpha = (float*)malloc((block_size+1)*8*sizeof(float));
Lext_input = (float*)malloc((block_size+1)*sizeof(float));
channel_x = (float*)malloc((block_size+1)*sizeof(float));
channel_y = (float*)malloc((block_size+1)*sizeof(float));

channel_x_before = (float*)malloc(guard_size*sizeof(float));
channel_y_before = (float*)malloc(guard_size*sizeof(float));
Lext_before = (float*)malloc(guard_size*sizeof(float));
channel_x_after = (float*)malloc(guard_size*sizeof(float));
channel_y_after = (float*)malloc(guard_size*sizeof(float));
Lext_after = (float*)malloc(guard_size*sizeof(float));


float beta[8], temp_beta[8];
float start_alpha[8], start_beta[8];
float gamma_0, gamma_1;
float alpha_0, alpha_1;
float beta_0, beta_1;
float splus, sminus, lambda_0, lambda_1;
float Lexternal;
int i, j;
int block_end;
//************Fetching channel and Lext values for decoding*****************
block_end = (figment == blocks-1) ? block_size : block_size+1;
for(i=0; i<block_end; i++)
{
	channel_y[i] = outbits_y[figment*block_size+i];
}
if(which_decoder == 1)
{
	for(i=0; i<block_end; i++)
	{
		Lext_input[i] = Lext_in[figment*block_size+i];
		channel_x[i] = outbits_x[figment*block_size+i];
	}
}
else if(which_decoder == 2)
{
	for(i=0; i<block_end; i++)
	{
		Lext_input[i] = Lext_in[inv_permutation_bits[figment*block_size+i]];
		channel_x[i] = outbits_x[inv_permutation_bits[figment*block_size+i]];
	}
}
//************Fetching channel and Lext values for decoding*****************

//**********************Fetching values for performing guarding**********************
//**********************Before the window fetching**********************
if(figment != 0)
{
	for(i=0; i<guard_size; i++)
	{
		channel_y_before[i] = outbits_y[figment*block_size - guard_size + i];
	}
	if(which_decoder == 1)
	{
		for(i=0; i<guard_size; i++)
		{
			channel_x_before[i] = outbits_x[figment*block_size - guard_size + i];
			Lext_before[i] = Lext_in[figment*block_size - guard_size + i];
		}
	}
	else if(which_decoder == 2)
	{
		for(i=0; i<guard_size; i++)
		{
			channel_x_before[i] = outbits_x[inv_permutation_bits[figment*block_size - guard_size + i]];
			Lext_before[i] = Lext_in[inv_permutation_bits[figment*block_size - guard_size + i]];
		}
	}
}
//**********************Before the window fetching**********************

//**********************After the window fetching**********************
if(figment != blocks-1)
{
	for(i=0; i<guard_size; i++)
	{
		channel_y_after[i] = outbits_y[(figment+1)*block_size +1 + i];
	}
	if(which_decoder == 1)
	{
		for(i=0; i<guard_size; i++)
		{
			channel_x_after[i] = outbits_x[(figment+1)*block_size +1 + i];
			Lext_after[i] = Lext_in[(figment+1)*block_size +1 + i];			
		}
	}
	else if(which_decoder == 2)
	{
		for(i=0; i<guard_size; i++)
		{
			channel_x_after[i] = outbits_x[inv_permutation_bits[(figment+1)*block_size +1 + i]];
			Lext_after[i] = Lext_in[inv_permutation_bits[(figment+1)*block_size +1 + i]];
			
		}
	}
}
//**********************After the window fetching**********************
//**********************Fetching values for performing guarding**********************


p_guard_alpha(start_alpha, read_ag, figment, guard_size, channel_x_before, channel_y_before, Lext_before, LC, guarding_type);
p_guard_beta(start_beta, read_bg, figment, guard_size, channel_x_after, channel_y_after, Lext_after, LC, blocks, which_decoder, trellis_term, guarding_type);

for(j=0; j<8; j++)
{
	alpha[j] = start_alpha[j];
	beta[j] = start_beta[j];
}


//*******************************************Alpha evaluation**********************************************
for(i=0; i<block_size; i++)
{
	for(j=0; j<8; j++)
	{
		gamma_0 = (0.5f)*(Lext_input[i] + LC*channel_x[i])*(-1) + (0.5f)*LC*channel_y[i]*cpu_alpha_encbit_0[j];
		gamma_1 = (0.5f)*(Lext_input[i] + LC*channel_x[i])*(1) + (0.5f)*LC*channel_y[i]*cpu_alpha_encbit_1[j];
		alpha_0 = alpha[i*8 + cpu_alpha_state_0[j] ] + gamma_0;
		alpha_1 = alpha[i*8 + cpu_alpha_state_1[j] ] + gamma_1;
		alpha[(i+1)*8 + j] = maxf(alpha_0, alpha_1);
	}
}
//***************************Writing end alpha for use in the next iteration***************************
if(guarding_type == 1)
{
	for(j=0; j<8; j++)
	{
		write_ag[figment*8+j] = alpha[block_size*8+j];
	}
}
else if(guarding_type == 3)
{
	for(j=0; j<8; j++)
	{
		write_ag[figment*8+j] = alpha[(block_size-guard_size)*8+j];
	}
}
//***************************Writing end alpha for use in the next iteration***************************

//*******************************************Alpha evaluation**********************************************
/************Print alpha test block*******************
FILE *fp1, *fp2, *fp3, *fp4;
fp1 = fopen("alpha_check_1.dat", "w");
fp2 = fopen("beta_check_1.dat", "w");
fp3 = fopen("Lext_check_1.dat", "w");
fp4 = fopen("Full_Lext_check_1.dat", "w");
for(i=0; i<block_size; i++)
{
	for(j=0; j<8; j++)
	{
		fprintf(fp1, "%f\t", alpha[(i+1)*8+j]);
	}
	fprintf(fp1, "\n");
}
fclose(fp1);
************Print alpha test block*******************/


//*******************************************Beta and Lext evaluation**********************************************
for(i=block_size-1; i>=0; i--)
{
	//***************************************Beta evaluation***********************************************
	if((figment == blocks-1) && (i == block_size-1))
	{
		for(j=0; j<8; j++)
		{
			beta[j] = EQUAL_GUARD_VALUE;
			temp_beta[j] = EQUAL_GUARD_VALUE;
		}
	}
	else
	{
		for(j=0; j<8; j++)
		{
			gamma_0 = (0.5f)*(Lext_input[i+1] + LC*channel_x[i+1])*(-1) + (0.5f)*LC*channel_y[i+1]*cpu_beta_encbit_0[j];
			gamma_1 = (0.5f)*(Lext_input[i+1] + LC*channel_x[i+1])*(1) + (0.5f)*LC*channel_y[i+1]*cpu_beta_encbit_1[j];
			beta_0 = beta[cpu_beta_state_0[j] ] + gamma_0;
			beta_1 = beta[cpu_beta_state_1[j] ] + gamma_1;
			temp_beta[j] = maxf(beta_0, beta_1);
		}	
		for(j=0; j<8; j++)
		{
			beta[j] = temp_beta[j];
		}
	}
	//***********************Writing beta for initialisation in the next iteration*************************
	if(guarding_type == 1)
	{
		if(i == 0)
		{
			for(j=0; j<8; j++)
			{
				write_bg[figment*8+j] = beta[j];
			}
		}
	}
	else if(guarding_type == 3)
	{
		if(i == guard_size)
		{
			for(j=0; j<8; j++)
			{
				write_bg[figment*8+j] = beta[j];
			}
		}
	}
	//***********************Writing beta for initialisation in the next iteration*************************
	/********Print beta block**********
	for(j=0; j<8; j++)
	{
		fprintf(fp2,"%f\t", beta[j]);
	}
	fprintf(fp2, "\n");
	********Print beta block**********/
	//***************************************Beta evaluation***********************************************

	//***************************************Lexternal evaluation******************************************	
	sminus = 0;
	splus =  0;
	for(j=0; j<8; j++)
	{
		gamma_0 = (0.5f)*(Lext_input[i] + LC*channel_x[i])*(-1) + (0.5f)*LC*channel_y[i]*cpu_beta_encbit_0[j];
		gamma_1 = (0.5f)*(Lext_input[i] + LC*channel_x[i])*(1) + (0.5f)*LC*channel_y[i]*cpu_beta_encbit_1[j];
		lambda_0 = alpha[i*8+j] + beta[cpu_beta_state_0[j]] + gamma_0;
		lambda_1 = alpha[i*8+j] + beta[cpu_beta_state_1[j]] + gamma_1;
		sminus = maxf(sminus, lambda_0);
		splus = maxf(splus, lambda_1);
	}
	Lexternal = splus - sminus - LC*channel_x[i]- Lext_input[i];
	/*************************************************Mega Lext check*****************************************************************
	fprintf(fp4, "%f\t%f\t%f\t%f\t%f\t%f\t%f\n", channel_x[i], Lexternal,sminus, splus,splus - sminus, LC*channel_x[i], Lext_input[i]);
	*************************************************Mega Lext check*****************************************************************/
	//***************************************Lexternal evaluation******************************************
	if(which_decoder == 1)
	{
		Lext_out[figment*block_size + i] = Lexternal;
	}
	else if(which_decoder == 2)
	{
		Lext_out[inv_permutation_bits[figment*block_size + i]] = Lexternal;
	}
}
//*******************************************Beta and Lext evaluation**********************************************
/************Printing Lexternal*****************
for(i=0; i<DATASIZE; i++)
{
	fprintf(fp3, "%f\n", Lext_out[i]);
}
fclose(fp2);
fclose(fp3);
fclose(fp4);
************Printing Lexternal*****************/
free(alpha);
free(channel_x);
free(channel_y);
free(Lext_input);
}



void p_guard_alpha(float *start_alpha, float *guard_alpha, int figment, int guard_size, float *channel_x_before, float *channel_y_before, float *Lext_before, float LC, int guarding_type)
{
float maxf(float a, float b);
int i,j,k;
float gamma_0, gamma_1, alpha_0, alpha_1;
float temp_alpha[8];
int cpu_alpha_state_0[8] 	= 	{0, 3, 4, 7, 1, 2, 5, 6};
int cpu_alpha_state_1[8] 	= 	{1, 2, 5, 6, 0, 3, 4, 7};
int cpu_alpha_encbit_0[8] 	= 	{-1, 1, 1, -1, -1, 1, 1, -1};
int cpu_alpha_encbit_1[8] 	= 	{1, -1, -1, 1, 1, -1, -1, 1};
if(figment == 0)
{
	start_alpha[0] = 0;
	for(i=1; i<8; i++)
	{
		start_alpha[i] = MINUS_INFINITY;
	}
		
}
else
{
	if(guarding_type == 1)
	{
		for(i=0; i<8; i++)
		{
			start_alpha[i] = guard_alpha[(figment-1)*8 + i];
		}
	
	}
	else if(guarding_type == 2 || guarding_type == 3)
	{
		if(guarding_type == 2)
		{
			for(j=0; j<8; j++)
			{
				start_alpha[j] = EQUAL_GUARD_VALUE;
			}
		}
		else if(guarding_type == 3)
		{
			for(j=0; j<8; j++)
			{
				start_alpha[j] = guard_alpha[(figment-1)*8 + j];
			}
		}
		for(k=0; k<guard_size; k++)
		{
			for(j=0; j<8; j++)
			{
				gamma_0 = (0.5f)*(Lext_before[k] + LC*channel_x_before[k])*(-1) + (0.5f)*LC*channel_y_before[k]*cpu_alpha_encbit_0[j];
				gamma_1 = (0.5f)*(Lext_before[k] + LC*channel_x_before[k])*(1) + (0.5f)*LC*channel_y_before[k]*cpu_alpha_encbit_1[j];
				alpha_0 = start_alpha[cpu_alpha_state_0[j] ] + gamma_0;
				alpha_1 = start_alpha[cpu_alpha_state_1[j] ] + gamma_1;
				temp_alpha[j] = maxf(alpha_0, alpha_1);
			}
			for(j=0; j<8; j++)
			{
				start_alpha[j] = temp_alpha[j];
			}
		}
	}
}
}

void p_guard_beta(float *start_beta, float *guard_beta, int figment, int guard_size, float *channel_x_after, float *channel_y_after, float *Lext_after, float LC, int blocks, int which_decoder, int trellis_term, int guarding_type)
{
float maxf(float a, float b);
int i,j,k;
float gamma_0, gamma_1, beta_0, beta_1;
float temp_beta[8];
int cpu_beta_state_0[8] 	= 	{0, 4, 5, 1, 2, 6, 7, 3};
int cpu_beta_state_1[8] 	= 	{4, 0, 1, 5, 6, 2, 3, 7};
int cpu_beta_encbit_0[8] 	= 	{-1, -1, 1, 1, 1, 1, -1, -1};
int cpu_beta_encbit_1[8] 	= 	{1, 1, -1, -1, -1, -1, 1, 1};
if(figment == blocks-1)
{
	if(which_decoder == 1 && trellis_term == 1)
	{
		start_beta[0] = 0;
		for(i=1; i<8; i++)
		{
			start_beta[i] = MINUS_INFINITY;
		}
	}
	else
	{
		for(i=0; i<8; i++)
		{
			start_beta[i] = EQUAL_GUARD_VALUE;
		}
	}	
}
else
{
	if(guarding_type == 1)
	{
		
		for(i=0; i<8; i++)
		{
			start_beta[i] = guard_beta[(figment+1)*8 + i];
		}
		
	}
	else if(guarding_type == 2 || guarding_type == 3)
	{
		if(guarding_type == 2)
		{
			for(j=0; j<8; j++)
			{
				start_beta[j] = EQUAL_GUARD_VALUE;
			}
		}
		else if(guarding_type == 3)
		{
			for(j=0; j<8; j++)
			{
				start_beta[j] = guard_beta[(figment+1)*8 + j];
			}
		}
		for(k=guard_size-1; k>=0; k--)
		{
			for(j=0; j<8; j++)
			{
				gamma_0 = (0.5f)*(Lext_after[k] + LC*channel_x_after[k])*(-1) + (0.5f)*LC*channel_y_after[k]*cpu_beta_encbit_0[j];
				gamma_1 = (0.5f)*(Lext_after[k] + LC*channel_x_after[k])*(1) + (0.5f)*LC*channel_y_after[k]*cpu_beta_encbit_1[j];
				beta_0 = start_beta[cpu_beta_state_0[j] ] + gamma_0;
				beta_1 = start_beta[cpu_beta_state_1[j] ] + gamma_1;
				temp_beta[j] = maxf(beta_0, beta_1);
			}	
			for(j=0; j<8; j++)
			{
				start_beta[j] = temp_beta[j];
			}
		}	
	}
}
}


