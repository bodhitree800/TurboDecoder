#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include "hashdefined.h"

void just_decoder_parallel_close(float snr, int iter_num, int blocks, int guard_size, float *outbits_x, float *outbits_y1, float *outbits_y2, int *decisionbits, int trellis_term, int guarding_type)
{
void result(float LC, float *outbits_x, int *decisionbits, float *Lext12, float *Lext21);
void parallel_decoder_close(int blocks, int guard_size, float *Lext_in, float *Lext_out, float *outbits_x, float *outbits_y,  float LC, int which_decoder);
float Lext12[DATASIZE], Lext21[DATASIZE];
extern int inv_permutation_bits[DATASIZE];
float LC;
int iter;
int i;
LC = (4.0/3.0)*pow(10,snr/10.0);
for(i=0; i<DATASIZE; i++)
{
	Lext21[i] = 0;
	Lext12[i] = 0;
}
for(iter =1; iter<= iter_num; iter++)
{
	parallel_decoder_close(blocks, guard_size, Lext21, Lext12, outbits_x, outbits_y1, LC, 1);
	parallel_decoder_close(blocks, guard_size, Lext12, Lext21, outbits_x, outbits_y2, LC, 2);	
}
/********Lext12 and Lext21 check block**************
FILE *fp;
int pi;
fp = fopen("Lext_host.dat", "w");
for(pi=0; pi<DATASIZE; pi++)
{
	fprintf(fp, "%f\t%f\t%f\t%f\n", LC*outbits_x[pi], Lext12[pi], Lext21[pi], LC*outbits_x[pi] + Lext12[pi] + Lext21[pi]);
}
fclose(fp);
********Lext12 and Lext21 check block**************/
result(LC, outbits_x, decisionbits, Lext12, Lext21);
}


void parallel_decoder_close(int blocks, int guard_size, float *Lext_in, float *Lext_out, float *outbits_x, float *outbits_y,  float LC, int which_decoder)
{
void figment_decoder_close(int figment, int blocks, int guard_size, float *Lext_in, float *Lext_out, float *channel_x, float *channel_y,  float LC, int which_decoder);
int i;
//******************************************Using the figment decoder to do parallel decoding******************************************
for(i=0; i<blocks; i++)
{
	figment_decoder_close(i, blocks, guard_size, Lext_in, Lext_out, outbits_x, outbits_y, LC, which_decoder);
}
//******************************************Using the figment decoder to do parallel decoding******************************************
}

void figment_decoder_close(int figment, int blocks, int guard_size, float *Lext_in, float *Lext_out, float *outbits_x, float *outbits_y,  float LC, int which_decoder)
{
extern int inv_permutation_bits[DATASIZE];
float maxf(float a, float b);
void guard_alpha(float *start_alpha);
void guard_beta(float *start_beta, int which_decoder, int trellis_term);
int cpu_alpha_state_0[8] 	= 	{0, 3, 4, 7, 1, 2, 5, 6};
int cpu_alpha_state_1[8] 	= 	{1, 2, 5, 6, 0, 3, 4, 7};
int cpu_beta_state_0[8] 	= 	{0, 4, 5, 1, 2, 6, 7, 3};
int cpu_beta_state_1[8] 	= 	{4, 0, 1, 5, 6, 2, 3, 7};
int cpu_alpha_encbit_0[8] 	= 	{-1, 1, 1, -1, -1, 1, 1, -1};
int cpu_alpha_encbit_1[8] 	= 	{1, -1, -1, 1, 1, -1, -1, 1};
int cpu_beta_encbit_0[8] 	= 	{-1, -1, 1, 1, 1, 1, -1, -1};
int cpu_beta_encbit_1[8] 	= 	{1, 1, -1, -1, -1, -1, 1, 1};
int block_size;
float *alpha, *Lext, *channel_x, *channel_y;
float *lambda_0, *lambda_1;
block_size = DATASIZE/blocks;
alpha = (float*)malloc((block_size+1)*8*sizeof(float));
Lext = (float*)malloc((block_size+1)*sizeof(float));
channel_x = (float*)malloc((block_size+1)*sizeof(float));
channel_y = (float*)malloc((block_size+1)*sizeof(float));
lambda_0 = (float*)malloc(8*8*sizeof(float));
lambda_1 = (float*)malloc(8*8*sizeof(float));

float beta[8], temp_beta[8];
float start_alpha[8], start_beta[8];
float gamma_0, gamma_1;
float alpha_0, alpha_1;
float beta_0, beta_1;
float Lext_0, Lext_1;
float Lexternal;
int i, j, k, l, tid, index;
int block_end;
block_end = (figment == blocks-1) ? block_size : block_size+1;
for(i=0; i<block_end; i++)
{
	channel_y[i] = outbits_y[figment*block_size+i];
}
if(which_decoder == 1)
{
	for(i=0; i<block_end; i++)
	{
		Lext[i] = Lext_in[figment*block_size+i];
		channel_x[i] = outbits_x[figment*block_size+i];
	}
}
else if(which_decoder == 2)
{
	for(i=0; i<block_end; i++)
	{
		Lext[i] = Lext_in[inv_permutation_bits[figment*block_size+i]];
		channel_x[i] = outbits_x[inv_permutation_bits[figment*block_size+i]];
	}
}

guard_alpha(start_alpha);
guard_beta(start_beta, 0, 1);

//*******************************************Alpha evaluation**********************************************
for(tid=0; tid<8; tid++)
{
	alpha[tid]=start_alpha[tid];
}
for(i=0; i<block_size; i++)
{
	for(tid=0; tid<8; tid++)
	{
		gamma_0 = (0.5f)*(Lext[i] + LC*channel_x[i])*(-1) + (0.5f)*LC*channel_y[i]*cpu_alpha_encbit_0[tid];
		gamma_1 = (0.5f)*(Lext[i] + LC*channel_x[i])*(1) + (0.5f)*LC*channel_y[i]*cpu_alpha_encbit_1[tid];
		alpha_0 = alpha[i*8 + cpu_alpha_state_0[tid] ] + gamma_0;
		alpha_1 = alpha[i*8 + cpu_alpha_state_1[tid] ] + gamma_1;
		alpha[(i+1)*8 + tid] = maxf(alpha_0, alpha_1);
	}
}
//*******************************************Alpha evaluation**********************************************
/************Print alpha test block*******************
int pj;
FILE *fp1, *fp2, *fp3, *fp4, *fp5, *fp6;
fp1 = fopen("alpha_check_2.dat", "w");
fp2 = fopen("beta_check_2.dat", "w");
fp3 = fopen("Lext_check_2.dat", "w");
fp4 = fopen("Full_Lext_check_2.dat", "w");
fp5 = fopen("lambda_0_check.dat", "w");
fp6 = fopen("lambda_1_check.dat",  "w");
for(i=0; i<block_size; i++)
{
	for(pj=0; pj<8; pj++)
	{
		fprintf(fp1, "%f\t", alpha[(i+1)*8+pj]);
	}
	fprintf(fp1, "\n");
}
fclose(fp1);
************Print alpha test block*******************/
//******************************Beta and Lext evaluation*******************************
for(tid=0; tid<8; tid++)
{
	beta[tid] = start_beta[tid];
}
for(k=block_size/8; k>0; k--)
{
	for(j=0; j<8; j++)
	{
		i = k*8-j-1;
		if((figment == blocks-1) && (i == block_size-1))
		{
			for(tid=0; tid<8; tid++)
			{
				temp_beta[tid] = EQUAL_GUARD_VALUE;
				beta[tid] = EQUAL_GUARD_VALUE;
			}
		}
		else
		{
			for(tid=0; tid<8; tid++)
			{
				//**********************************Beta evaluation****************************
				gamma_0 = (0.5f)*(LC*channel_x[i+1]+Lext[i+1])*(-1) + (0.5f)*LC*channel_y[i+1]*cpu_beta_encbit_0[tid];
				gamma_1 = (0.5f)*(LC*channel_x[i+1]+Lext[i+1])*(1) + (0.5f)*LC*channel_y[i+1]*cpu_beta_encbit_1[tid];
				beta_0 = beta[cpu_beta_state_0[tid]] + gamma_0;
				beta_1 = beta[cpu_beta_state_1[tid]] + gamma_1;
				temp_beta[tid] = maxf(beta_0, beta_1);
			}
			for(tid=0; tid<8; tid++)
			{
				beta[tid] = temp_beta[tid];
			}
		}
		for(tid=0; tid<8; tid++)
		{
			//**********************************Lambda evaluation**************************
			gamma_0 = (0.5f)*(LC*channel_x[i]+Lext[i])*(-1) + (0.5f)*LC*channel_y[i]*cpu_beta_encbit_0[tid];
			gamma_1 = (0.5f)*(LC*channel_x[i]+Lext[i])*(1) + (0.5f)*LC*channel_y[i]*cpu_beta_encbit_1[tid];
			lambda_0[j*8+tid] = alpha[i*8+tid] + temp_beta[cpu_beta_state_0[tid]] + gamma_0;
			lambda_1[j*8+tid] = alpha[i*8+tid] + temp_beta[cpu_beta_state_1[tid]] + gamma_1;
			//**********************************Lambda evaluation**************************/
		}
		/**************Print lambda block**************
		for(pj=0; pj<8; pj++)
		{
			fprintf(fp5, "%f\t", lambda_0[j*8+pj]);
			fprintf(fp6, "%f\t", lambda_1[j*8+pj]);
		}
		fprintf(fp5, "\n");
		fprintf(fp6, "\n");
		**************Print lambda block**************/	
		/********Print beta block**********
		for(pj=0; pj<8; pj++)
		{
			fprintf(fp2,"%f\t", beta[pj]);
		}
		fprintf(fp2, "\n");
		********Print beta block************/
	}
	for(tid=0; tid<8; tid++)
	{
		//****************************Lexternal evaluation****************************
		Lext_0 = lambda_0[tid*8];
		Lext_1 = lambda_1[tid*8];
		for(l=1; l<=7; l++)
		{
				Lext_0 = maxf(Lext_0, lambda_0[tid*8 + l]);
				Lext_1 = maxf(Lext_1, lambda_1[tid*8 + l]);
		}
		Lexternal = (Lext_1 - Lext_0) - LC*channel_x[k*8 - tid - 1] - Lext[k*8 - tid - 1];
		/****************Full Lext print check**********************
		fprintf(fp4, "%f\t%f\t%f\t%f\t%f\n", channel_x[i+7-tid], Lexternal,Lext_0, Lext_1, Lext[i+7-tid]);
		****************Full Lext print check**********************/
		index = figment*block_size + k*8 - tid - 1;
		if(which_decoder == 1)
		{
			Lext_out[index] = Lexternal;
		}
		else if(which_decoder == 2)
		{
			Lext_out[inv_permutation_bits[index]] = Lexternal;
		}
		//****************************Lexternal evaluation****************************/	
	}	
}
/************Printing Lexternal*****************
for(i=0; i<DATASIZE; i++)
{
	fprintf(fp3, "%f\n", Lext_out[i]);
}
fclose(fp2);
fclose(fp3);
fclose(fp4);
fclose(fp5);
fclose(fp6);
************Printing Lexternal*****************/
free(alpha);
free(Lext);
free(channel_x);
free(channel_y);
free(lambda_0);
free(lambda_1);
}
