#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>
#include "hashdefined.h"

void encoder_and_noise(int numb_of_bits, int trellis_termination, float snr, int noise_enable)
//Output of this is a file by name "bits_to_be_decoded.dat"
{
void bit_generator(int numb_of_bits, int trellis_termination);
void just_encoder(int numb_of_bits);
void awgnchannel(int numb_of_bits, float snr, int noise_enable);
bit_generator(numb_of_bits, trellis_termination);
just_encoder(numb_of_bits);
awgnchannel(numb_of_bits, snr, noise_enable);
}

void bit_generator(int numb_of_bits, int trellis_termination)
//Generates random bits and stores them in the file named "input_bits.dat"
//Takes as input number of bits to generate
//Expects '0' or '1' as the trellis termination bit
//'0' --> No trellis termination
//'1' --> Trellis termination is present
//Encoding is necessary to do the trellis termination
{
int next_state(int current_state, int input);
int i, k, numb = numb_of_bits/DATASIZE;
int input_bit;
int currentstate = 0;
int trellis_term_table[8][3] = {{0,0,0},{1,0,0},{1,1,0},{0,1,0},{0,1,1},{1,1,1},{1,0,1},{0,0,1}};
numb_of_bits = ((numb_of_bits+DATASIZE-1)/DATASIZE)*DATASIZE;
if(trellis_termination == 0 || trellis_termination == 1)
{
	FILE *fp;
	fp = fopen("input_bits.dat", "w");
	if(trellis_termination == 0)
	{
		srand((unsigned int)time((time_t *)NULL)); //modifying the seed for rand based on time
		//srand(time(NULL));
		for(i = 1; i <= numb_of_bits; i++)
		{
			fprintf(fp, "%d\n", rand() % 2);
		}
	}
	else if(trellis_termination == 1)
	{
		for(k=1; k<= numb; k++)
		{
			srand((unsigned int)time((time_t *)NULL));
			for(i=1; i<=DATASIZE-3; i++)
			{
				input_bit = rand() % 2;
				currentstate = next_state(currentstate, input_bit);
				fprintf(fp, "%d\n", input_bit);
			}
			fprintf(fp, "%d\n", trellis_term_table[currentstate][0]);
			fprintf(fp, "%d\n", trellis_term_table[currentstate][1]);
			fprintf(fp, "%d\n", trellis_term_table[currentstate][2]);
		}
	}
	fclose(fp);
}
else
{
	printf("Improper input to decide upon trellis termination\n");
	printf("Exiting abruptly\n");
	exit(1);
}	
}


void just_encoder(int numb_of_bits)
//encodes the bits stored in the file "input_bits.dat"
//Stores the encoded bits in a new file "encoded_bits.dat"
{
int interleaver(int index);
int next_state(int current_state, int input);
int output(int current_state, int input);
FILE *fp, *gp;
int inputbits[DATASIZE], deinter_bits[DATASIZE], encbits_1[DATASIZE], encbits_2[DATASIZE];
int currentstate;
int i,k, numb;
fp = fopen("input_bits.dat","r");
gp = fopen("encoded_bits.dat","w");
numb_of_bits = ((numb_of_bits+DATASIZE-1)/DATASIZE)*DATASIZE;
numb = numb_of_bits/DATASIZE;
for(k = 0; k<numb; k++)
{
	for(i = 0; i < DATASIZE; i++)
	{
		fscanf(fp, "%d\n", &inputbits[i]);
		deinter_bits[interleaver(i)] = inputbits[i];
	}
	//*****************Encoder 1********************
	currentstate = 0;
	for(i = 0; i<DATASIZE; i++)
	{
		encbits_1[i] = output(currentstate, inputbits[i]);
		currentstate = next_state(currentstate, inputbits[i]);
	}
	//*****************Encoder 1********************
	//*****************Encoder 2********************
	currentstate = 0;
	for( i = 0; i<DATASIZE; i++)
	{
		encbits_2[i] = output(currentstate, deinter_bits[i]);
		currentstate = next_state(currentstate, deinter_bits[i]);
	}
	//*****************Encoder 2********************
	for(i = 0; i < DATASIZE; i++)
	{
		fprintf(gp, "%d\t%d\t%d\n", inputbits[i]==0 ? -1 : 1, encbits_1[i], encbits_2[i]);
	}
}
fclose(fp);
fclose(gp);
}



void awgnchannel(int numb_of_bits, float snr, int noise_enable)
//Takes as input the encoded bits file by name encoded_bits.dat
//Adds noise to the encoded bits of that file
//Takes as input the snr of the noise to be added
{
float gasdev(long *idum);
int bit, enc1_bit, enc2_bit, i;
float outbit_x, outbit_y1, outbit_y2;
FILE *fp, *gp;
fp = fopen("encoded_bits.dat", "r");
gp = fopen("channel_out.dat", "w");
long int seedvalue = ((unsigned int)time((time_t *)NULL));//seed needs to be negative for ran1() which gasdev() calls
seedvalue = -seedvalue;
float sigma, rate = 1.0/3.0;
sigma = pow(10,-snr/20)/sqrt(2*rate); //The SNR value input is in db
numb_of_bits = ((numb_of_bits+DATASIZE-1)/DATASIZE)*DATASIZE;
if(noise_enable == 1)
{
	for(i=0; i<numb_of_bits; i++)
	{
		//***********************with noise****************************
		fscanf(fp, "%d\t%d\t%d\n", &bit, &enc1_bit, &enc2_bit);
		outbit_x = bit + sigma*gasdev(&seedvalue);
		outbit_y1 = enc1_bit + sigma*gasdev(&seedvalue);
		outbit_y2 = enc2_bit + sigma*gasdev(&seedvalue);
		fprintf(gp, "%f\t%f\t%f\n", outbit_x, outbit_y1, outbit_y2);
		//***********************with noise****************************/
	}
}
else if(noise_enable == 0)
{
	for(i=0; i<numb_of_bits; i++)
	{
		//***********************without noise****************************
		fscanf(fp, "%d\t%d\t%d\n", &bit, &enc1_bit, &enc2_bit);
		outbit_x = bit;
		outbit_y1 = enc1_bit;
		outbit_y2 = enc2_bit;
		fprintf(gp, "%f\t%f\t%f\n", outbit_x, outbit_y1, outbit_y2);
		//***********************without noise****************************/
	}
}
fclose(fp);
fclose(gp);
}

void permuter_bits()
{
int interleaver(int n);
extern int inv_permutation_bits[DATASIZE];
int k;
for(k=0; k<DATASIZE; k++)
{
	inv_permutation_bits[(((1+6*k)%DATASIZE)*k)%DATASIZE] = k;
}
/**********************Deinterleaver check block****************
FILE *fp;
fp = fopen("deinter.bits.dat", "w");
for(k=0; k<DATASIZE; k++)
{

	fprintf(fp,"%d\t%d\t%d\n",k,interleaver(k), inv_permutation_bits[k]);
}
**********************Deinterleaver check block****************/
}

int interleaver(int n)
{
return (((1+6*n)%DATASIZE)*n)%DATASIZE;
}


int deinterleaver(int index)
{
//performs de-interleaving using the already created permutation_bits array
extern int inv_permutation_bits[DATASIZE];
return inv_permutation_bits[index];
}
