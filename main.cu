#include "hip/hip_runtime.h"
// #include "cuPrintf.cu"
#include "decoderlog.cu"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>
#include "hashdefined.h"
int inv_permutation_bits[DATASIZE];
main()
{
	void decoder_types();
	void simulations();
	decoder_types();
	// simulations();
	system("rm decoder_output.dat encoded_bits.dat channel_out.dat input_bits.dat");
}

void simulations()
{
	void simulation_iter(int numb_of_bits, int decoder_type, int guarding_type, int guard_size, int trellis_term_enable, int only_plotting);
	void simulation_guard(int numb_of_bits, int decoder_type, int iter, int trellis_term_enable, int only_plotting);
	void simulation_blocks(int numb_of_bits, int decoder_type, int iter, int guarding_type, int guard_size, int trellis_term_enable, int only_plotting);
	int numb_of_bits = 1e8;
	int decoder_type = 4;
	int guarding_type = 3;
	int guard_size = 8;
	int iter = 5;
	int trellis_term_enable = 0;
	simulation_iter(numb_of_bits, decoder_type, guarding_type, guard_size, trellis_term_enable, 0);
	simulation_guard(numb_of_bits, decoder_type, iter, trellis_term_enable, 0);
	simulation_blocks(numb_of_bits, decoder_type, iter, guarding_type, guard_size, trellis_term_enable, 0);
}

void decoder_types()
{
	void permuter_bits();
	void encoder_and_noise(int numb_of_bits, int trellis_termination, float snr, int noise_enable);
	void decode_and_analyse(int numb_of_bits, float snr, int iter_num, int blocks, int guard_size, int trellis_term_enable, int decoder_kind, int guarding_type, int *bit_error_count, int *frame_error_count);
	float snr;
	int numb_of_bits, iter, blocks, guard_size;
	int trellis_term_enable, noise_enable;
	int bit_error_count, frame_error_count;
	trellis_term_enable = 0;
	noise_enable = 1;
	iter = 5;
	snr = 1.5;
	numb_of_bits = 1e7;
	blocks = NO_OF_BLOCKS;
	guard_size = 5;
	permuter_bits();
	encoder_and_noise(numb_of_bits, trellis_term_enable, snr, noise_enable);
	decode_and_analyse(numb_of_bits, snr, iter, blocks, guard_size, trellis_term_enable, 4, 1, &bit_error_count, &frame_error_count);
}

void decode_and_analyse(int numb_of_bits, float snr, int iter_num, int blocks, int guard_size, int trellis_term_enable, int decoder_kind, int guarding_type, int *bit_error_count, int *frame_error_count)
{
	void full_decoder(int numb_of_bits, float snr, int iter_num, int blocks, int guard_size, int trellis_term, int decoder_kind, int guarding_type);
	void analysis(int numb_of_bits, int *bit_error_count, int *frame_error_count);
	full_decoder(numb_of_bits, snr, iter_num, blocks, guard_size, trellis_term_enable, decoder_kind, guarding_type);
	analysis(numb_of_bits, bit_error_count, frame_error_count);
}

void full_decoder(int numb_of_bits, float snr, int iter_num, int blocks, int guard_size, int trellis_term, int decoder_kind, int guarding_type)
{
	void just_decoder_basic(float snr, int iter_num, float *outbits_x, float *outbits_y1, float *outbits_y2, int *decision_bits, int trellis_term);
	void just_decoder_parallel(float snr, int iter_num, int blocks, int guard_size, float *outbits_x, float *outbits_y1, float *outbits_y2, int *decision_bits, int trellis_term, int guarding_type);
	void just_decoder_parallel_close(float snr, int iter_num, int blocks, int guard_size, float *outbits_x, float *outbits_y1, float *outbits_y2, int *decisionbits, int trellis_term, int guarding_type);
	void just_decoder_gpu(float snr, int iter_num, int blocks, int guard_size, float *outbits_x, float *outbits_y1, float *outbits_y2, int *decision_bits, int trellis_term, int guarding_type);
	clock_t start = clock(), hold1, hold2;
	//*************Loading up decoder inputs to memory**************
	numb_of_bits = ((numb_of_bits + DATASIZE - 1) / DATASIZE) * DATASIZE;
	FILE *fp, *gp;
	fp = fopen("channel_out.dat", "r");
	gp = fopen("decoder_output.dat", "w");
	float *mega_outbits_x, *mega_outbits_y1, *mega_outbits_y2;
	int *mega_decision_bits;
	mega_outbits_x = (float *)malloc((numb_of_bits) * sizeof(float));
	mega_outbits_y1 = (float *)malloc((numb_of_bits) * sizeof(float));
	mega_outbits_y2 = (float *)malloc((numb_of_bits) * sizeof(float));
	mega_decision_bits = (int *)malloc((numb_of_bits) * sizeof(int));
	int numb;
	int i;
	for (i = 0; i < numb_of_bits; i++)
	{
		fscanf(fp, "%f\t%f\t%f\n", &mega_outbits_x[i], &mega_outbits_y1[i], &mega_outbits_y2[i]);
	}
	fclose(fp);
	//*************Done Loading up decoder inputs to memory***************
	// printf("Time elapsed for loading into memory: %f\n", ((double)clock() - start) / CLOCKS_PER_SEC);
	hold1 = clock();
	//**************************Using cudahostalloc so speed up cudamemcpy**************************
	float *outbits_x, *outbits_y1, *outbits_y2;
	int *decisionbits;
	hipHostAlloc((void **)&outbits_x, DATASIZE * sizeof(float), hipHostMallocDefault);
	hipHostAlloc((void **)&outbits_y1, DATASIZE * sizeof(float), hipHostMallocDefault);
	hipHostAlloc((void **)&outbits_y2, DATASIZE * sizeof(float), hipHostMallocDefault);
	hipHostAlloc((void **)&decisionbits, DATASIZE * sizeof(int), hipHostMallocDefault);
	//**************************Using cudahostalloc so speed up cudamemcpy**************************
	for (numb = 0; numb < numb_of_bits / DATASIZE; numb++)
	{
		for (i = 0; i < DATASIZE; i++)
		{
			outbits_x[i] = mega_outbits_x[numb * DATASIZE + i];
			outbits_y1[i] = mega_outbits_y1[numb * DATASIZE + i];
			outbits_y2[i] = mega_outbits_y2[numb * DATASIZE + i];
		}
		//***********DATASIZE decoder******************
		switch (decoder_kind)
		{
		case 1:
		{
			just_decoder_basic(snr, iter_num, outbits_x, outbits_y1, outbits_y2, decisionbits, trellis_term);
			break;
		}
		case 2:
		{
			just_decoder_parallel(snr, iter_num, blocks, guard_size, outbits_x, outbits_y1, outbits_y2, decisionbits, trellis_term, guarding_type);
			break;
		}
		case 3:
		{
			just_decoder_parallel_close(snr, iter_num, blocks, guard_size, outbits_x, outbits_y1, outbits_y2, decisionbits, trellis_term, guarding_type);
			break;
		}
		case 4:
		{
			just_decoder_gpu(snr, iter_num, blocks, guard_size, outbits_x, outbits_y1, outbits_y2, decisionbits, trellis_term, guarding_type);
			break;
		}
		default:
		{
			printf("Wrong value give to decoder_kind\ncheck, exiting abruptly\n");
			exit(1);
		}
		}
		//***********DATASIZE decoder******************
		for (i = 0; i < DATASIZE; i++)
		{
			mega_decision_bits[numb * DATASIZE + i] = decisionbits[i];
		}
	}
	hold2 = clock();
	printf("Speed of decoding = %d Kbits/sec\n", (int)((numb_of_bits / ((double)(hold2 - hold1) / CLOCKS_PER_SEC)) / 1e3));
	// printf("Time elapsed for decoding: %f\n", ((double)clock() - hold1) / CLOCKS_PER_SEC);
	// printf("%d\n",(int)((numb_of_bits/((double)(hold2-hold1)/CLOCKS_PER_SEC))/1e3));
	//**************************Freeing up cudahostalloc allocated memory**************************
	hipHostFree(outbits_x);
	hipHostFree(outbits_y1);
	hipHostFree(outbits_y2);
	hipHostFree(decisionbits);
	//**************************Freeing up cudahostalloc allocated memory**************************
	//**************Storing back the decoded bits to file*******************
	for (i = 0; i < numb_of_bits; i++)
	{
		fprintf(gp, "%d\n", mega_decision_bits[i]);
	}
	free(mega_outbits_x);
	free(mega_outbits_y1);
	free(mega_outbits_y2);
	free(mega_decision_bits);
	fclose(gp);
	//**************Done storing back the decoded bits to file**************
	// printf("Time elapsed for writing to file: %f\n", ((double)clock() - hold2) / CLOCKS_PER_SEC);
}

void analysis(int numb_of_bits, int *bit_error_count, int *frame_error_count)
{
	FILE *fp, *gp;
	fp = fopen("input_bits.dat", "r");
	gp = fopen("decoder_output.dat", "r");
	int i, k;
	int actual_bit, decision_bit;
	int frame_error_flag, no_of_frames;
	numb_of_bits = ((numb_of_bits + DATASIZE - 1) / DATASIZE) * DATASIZE;
	no_of_frames = numb_of_bits / DATASIZE;
	*bit_error_count = 0;
	*frame_error_count = 0;
	for (k = 0; k < no_of_frames; k++)
	{
		frame_error_flag = 0;
		for (i = 0; i < DATASIZE; i++)
		{
			fscanf(fp, "%d\n", &actual_bit);
			fscanf(gp, "%d\n", &decision_bit);
			if (actual_bit != decision_bit)
			{
				(*bit_error_count)++;
				frame_error_flag = 1;
				// printf("%d----%d\n", i%DATASIZE, i/DATASIZE);
			}
		}
		if (frame_error_flag == 1)
		{
			(*frame_error_count)++;
		}
	}
	fclose(fp);
	fclose(gp);
	printf("Bit_Error_count = %d\nFrame_Error_count=%d\n", *bit_error_count, *frame_error_count);
	printf("BER = %g\nFER = %f\n", ((*bit_error_count) * 1.0) / numb_of_bits, ((*frame_error_count) * 1.0) / no_of_frames);
	// printf("percentage correct = %f\n", ((numb_of_bits-error_count)*100.0)/numb_of_bits);
}
