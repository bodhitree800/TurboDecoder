#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include "hashdefined.h"
void ber_plotting_1(int size, int *iter_array, int decoder_type, int guarding_type, int guard_size)
{
//Calling gnuplot and setting up various things in it
int i;
FILE *fp;
printf("Calling gnuplot\n");
fp = fopen("ber_gnucommands_1.txt","w");
fprintf(fp,"set term postscript eps enhanced color\n");
fprintf(fp,"set output \"BER_vs_SNR_plot_iter.ps\"\n");
fprintf(fp,"set xlabel \"SNR(in dB)\"\n");
fprintf(fp,"set ylabel \"BER(logscale)\"\n");
fprintf(fp,"set title \"BER vs SNR plot for different iterations\"\n");
switch(guarding_type)
{
	case 1 : {
			fprintf(fp,"set title \"Guarding type - only Previous value Initialisation\"\n");
		 	break;
		 }
	case 2 : {
			fprintf(fp,"set title \"Guarding type - only training window\"\tGuard size = %d\n", guard_size);
			break;
		 }
	case 3 : {
			fprintf(fp,"set title \"Guarding type - Previous value initialisation + training window, Guard size = %d\"\n", guard_size);
			break;
		 }
}
fprintf(fp,"set logscale y\n");
fprintf(fp,"plot");
for(i = 0; i<size-1; i++)
{
	fprintf(fp,"\"ber_vs_snr_iters_gnu.dat\" index %d:%d using 1:3 title \"Iter %d\" with lines,\\\n",i,i,iter_array[i]);
}
fprintf(fp,"\"ber_vs_snr_iters_gnu.dat\" index %d:%d using 1:3 title \"Iter %d\" with lines\n", size-1, size-1, iter_array[size-1]);
fclose(fp);
system("gnuplot ber_gnucommands_1.txt"); //using the script file gnucommands to plot
printf("Done plotting, go to the folder to see the plot\n");
}

void ber_plotting_3(int size, int *blocks_array, int decoder_type, int iter, int guarding_type, int guard_size)
{
//Calling gnuplot and setting up various things in it
int i;
FILE *fp;
printf("Calling gnuplot\n");
fp = fopen("ber_gnucommands_3.txt","w");
fprintf(fp,"set term postscript eps enhanced color\n");
fprintf(fp,"set output \"BER_vs_SNR_plot_block.ps\"\n");
fprintf(fp,"set xlabel \"SNR(in dB)\"\n");
fprintf(fp,"set ylabel \"BER(logscale)\"\n");
fprintf(fp,"set title \"BER vs SNR plot for different no. of blocks\"\n");
switch(guarding_type)
{
	case 1 : {
			fprintf(fp,"set title \"Guarding type - only Previous value Initialisation\"\n");
		 	break;
		 }
	case 2 : {
			fprintf(fp,"set title \"Guarding type - Only training window, Guard size = %d\"\n", guard_size);
			break;
		 }
	case 3 : {
			fprintf(fp,"set title \"Guarding type - Previous value initialisation + training window, Guard size = %d\"\n", guard_size);
			break;
		 }
}
fprintf(fp,"set logscale y\n");
fprintf(fp,"plot");
for(i = 0; i<size-1; i++)
{
	fprintf(fp,"\"ber_vs_snr_blocks_gnu.dat\" index %d:%d using 1:3 title \"# of blocks %d\" with lines,\\\n",i,i,blocks_array[i]);
}
fprintf(fp,"\"ber_vs_snr_blocks_gnu.dat\" index %d:%d using 1:3 title \"# of blocks %d\" with lines\n", size-1, size-1, blocks_array[size-1]);
fclose(fp);
system("gnuplot ber_gnucommands_3.txt"); //using the script file gnucommands to plot
printf("Done plotting, go to the folder to see the plot\n");
}

void ber_plotting_2(int size_2, int size_3, int *guard_size_2, int *guard_size_3, int decoder_type, int iter)
{
//Calling gnuploat and setting up various things in it
int i;
FILE *fp;
printf("Calling gnuplot\n");
fp = fopen("ber_gnucommands_2.txt","w");
fprintf(fp,"set term postscript eps enhanced color\n");
fprintf(fp,"set output \"BER_vs_SNR_plot_guard.ps\"\n");
fprintf(fp,"set xlabel \"SNR(in dB)\"\n");
fprintf(fp,"set ylabel \"BER(logscale)\"\n");
fprintf(fp,"set title \"BER vs SNR plot for different types of guarding, No. of iterations = %d\"\n", iter);
fprintf(fp,"set logscale y\n");
fprintf(fp,"plot");
fprintf(fp,"\"ber_vs_snr_guard_gnu.dat\" index 0:0 using 1:3 title \"Prev val init\" with lines,\\\n");
for(i = 1; i<size_2+1; i++)
{
	fprintf(fp,"\"ber_vs_snr_guard_gnu.dat\" index %d:%d using 1:3 title \"Only guard, size=%d\" with lines,\\\n",i,i,guard_size_2[i-1]);
}
for(i = size_2+1; i<size_3+size_2+1; i++)
{
	fprintf(fp,"\"ber_vs_snr_guard_gnu.dat\" index %d:%d using 1:3 title \"Prev val init and guard, size=%d\" with lines,\\\n",i,i,guard_size_3[i-1-size_2]);
}
fprintf(fp,"\"ber_vs_snr_guard_gnu.dat\" index %d:%d using 1:3 title \"Without guarding at all\" with lines\n", size_3+size_2+1, size_3+size_2+1);
//fprintf(fp,"\"ber_vs_snr_guard_gnu.dat\" index %d:%d using 1:3 title \"Max-log map with no parallelisation\" with lines\n", size_3+size_2+2, size_3+size_2+2);
fclose(fp);
system("gnuplot ber_gnucommands_2.txt"); //using the script file gnucommands to plot
printf("Done plotting, go to the folder to see the plot\n");
}

void fer_plotting_1(int size, int *iter_array, int decoder_type, int guarding_type, int guard_size)
{
//Calling gnuplot and setting up various things in it
int i;
FILE *fp;
printf("Calling gnuplot\n");
fp = fopen("fer_gnucommands_1.txt","w");
fprintf(fp,"set term postscript eps enhanced color\n");
fprintf(fp,"set output \"FER_vs_SNR_plot_iter.ps\"\n");
fprintf(fp,"set xlabel \"SNR(in dB)\"\n");
fprintf(fp,"set ylabel \"FER(logscale)\"\n");
fprintf(fp,"set title \"FER vs SNR plot for different iterations\"\n");
switch(guarding_type)
{
	case 1 : {
			fprintf(fp,"set title \"Guarding type - only Previous value Initialisation\"\n");
		 	break;
		 }
	case 2 : {
			fprintf(fp,"set title \"Guarding type - only training window\"\tGuard size = %d\n", guard_size);
			break;
		 }
	case 3 : {
			fprintf(fp,"set title \"Guarding type - Previous value initialisation + training window, Guard size = %d\"\n", guard_size);
			break;
		 }
}
fprintf(fp,"set logscale y\n");
fprintf(fp,"plot");
for(i = 0; i<size-1; i++)
{
	fprintf(fp,"\"fer_vs_snr_iters_gnu.dat\" index %d:%d using 1:3 title \"Iter %d\" with lines,\\\n",i,i,iter_array[i]);
}
fprintf(fp,"\"fer_vs_snr_iters_gnu.dat\" index %d:%d using 1:3 title \"Iter %d\" with lines\n", size-1, size-1, iter_array[size-1]);
fclose(fp);
system("gnuplot fer_gnucommands_1.txt"); //using the script file gnucommands to plot
printf("Done plotting, go to the folder to see the plot\n");
}




void fer_plotting_3(int size, int *blocks_array, int decoder_type, int iter, int guarding_type, int guard_size)
{
//Calling gnuplot and setting up various things in it
int i;
FILE *fp;
printf("Calling gnuplot\n");
fp = fopen("fer_gnucommands_3.txt","w");
fprintf(fp,"set term postscript eps enhanced color\n");
fprintf(fp,"set output \"FER_vs_SNR_plot_block.ps\"\n");
fprintf(fp,"set xlabel \"SNR(in dB)\"\n");
fprintf(fp,"set ylabel \"FER(logscale)\"\n");
fprintf(fp,"set title \"FER vs SNR plot for different no. of blocks\"\n");
switch(guarding_type)
{
	case 1 : {
			fprintf(fp,"set title \"Guarding type - only Previous value Initialisation\"\n");
		 	break;
		 }
	case 2 : {
			fprintf(fp,"set title \"Guarding type - Only training window, Guard size = %d\"\n", guard_size);
			break;
		 }
	case 3 : {
			fprintf(fp,"set title \"Guarding type - Previous value initialisation + training window, Guard size = %d\"\n", guard_size);
			break;
		 }
}
fprintf(fp,"set logscale y\n");
fprintf(fp,"plot");
for(i = 0; i<size-1; i++)
{
	fprintf(fp,"\"fer_vs_snr_blocks_gnu.dat\" index %d:%d using 1:3 title \"# of blocks %d\" with lines,\\\n",i,i,blocks_array[i]);
}
fprintf(fp,"\"fer_vs_snr_blocks_gnu.dat\" index %d:%d using 1:3 title \"# of blocks %d\" with lines\n", size-1, size-1, blocks_array[size-1]);
fclose(fp);
system("gnuplot fer_gnucommands_3.txt"); //using the script file gnucommands to plot
printf("Done plotting, go to the folder to see the plot\n");
}

void fer_plotting_2(int size_2, int size_3, int *guard_size_2, int *guard_size_3, int decoder_type, int iter)
{
//Calling gnuploat and setting up various things in it
int i;
FILE *fp;
printf("Calling gnuplot\n");
fp = fopen("fer_gnucommands_2.txt","w");
fprintf(fp,"set term postscript eps enhanced color\n");
fprintf(fp,"set output \"FER_vs_SNR_plot_guard.ps\"\n");
fprintf(fp,"set xlabel \"SNR(in dB)\"\n");
fprintf(fp,"set ylabel \"FER(logscale)\"\n");
fprintf(fp,"set title \"FER vs SNR plot for different types of guarding, No. of iterations = %d\"\n", iter);
fprintf(fp,"set logscale y\n");
fprintf(fp,"plot");
fprintf(fp,"\"fer_vs_snr_guard_gnu.dat\" index 0:0 using 1:3 title \"Prev val init\" with lines,\\\n");
for(i = 1; i<size_2+1; i++)
{
	fprintf(fp,"\"fer_vs_snr_guard_gnu.dat\" index %d:%d using 1:3 title \"Only guard, size=%d\" with lines,\\\n",i,i,guard_size_2[i-1]);
}
for(i = size_2+1; i<size_3+size_2+1; i++)
{
	fprintf(fp,"\"fer_vs_snr_guard_gnu.dat\" index %d:%d using 1:3 title \"Prev val init and guard, size=%d\" with lines,\\\n",i,i,guard_size_3[i-1-size_2]);
}
fprintf(fp,"\"fer_vs_snr_guard_gnu.dat\" index %d:%d using 1:3 title \"Without guarding at all\" with lines\n", size_3+size_2+1, size_3+size_2+1);
//fprintf(fp,"\"fer_vs_snr_guard_gnu.dat\" index %d:%d using 1:3 title \"Max-log map with no parallelisation\" with lines\n", size_3+size_2+2, size_3+size_2+2);
fclose(fp);
system("gnuplot fer_gnucommands_2.txt"); //using the script file gnucommands to plot
printf("Done plotting, go to the folder to see the plot\n");
}


