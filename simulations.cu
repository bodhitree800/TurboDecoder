#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include "hashdefined.h"
void simulation_iter(int numb_of_bits, int decoder_type, int guarding_type, int guard_size, int trellis_term_enable, int only_plotting)
{
	// Plots BER vs SNR in logscale for different iterations
	// Input information to be given is type of guarding, guard_size
	void permuter_bits();
	void encoder_and_noise(int numb_of_bits, int trellis_termination, float snr, int noise_enable);
	void decode_and_analyse(int numb_of_bits, float snr, int iter_num, int blocks, int guard_size, int trellis_term_enable, int decoder_kind, int guarding_type, int *bit_error_count, int *frame_error_count);
	void ber_plotting_1(int size, int *iter_array, int decoder_type, int guarding_type, int guard_size);
	void fer_plotting_1(int size, int *iter_array, int decoder_type, int guarding_type, int guard_size);
	float snr;
	float snr_start = 0.0;
	float snr_end = 2.005;
	float resolution = 0.1;
	int iter_array[10] = {1, 2, 3, 4, 5, 6, 7, 8, 9, 10};
	int size = 10;
	int i, k, bit_error_count, frame_error_count, numb_snrs = 0;
	int *bit_errordata, *frame_errordata;
	FILE *fp, *gp, *fpd, *gpd;
	numb_of_bits = ((numb_of_bits + DATASIZE - 1) / DATASIZE) * DATASIZE;
	int numb_of_frames = numb_of_bits / DATASIZE;
	if (only_plotting != 1)
	{
		fp = fopen("ber_vs_snr_iters.dat", "w");
		gp = fopen("ber_vs_snr_iters_linear.dat", "w");
		fpd = fopen("fer_vs_snr_iters.dat", "w");
		gpd = fopen("fer_vs_snr_iters_linear.dat", "w");
		permuter_bits();
		for (snr = snr_start; snr <= snr_end; snr += resolution)
		{
			encoder_and_noise(numb_of_bits, 0, snr, 1);
			fprintf(fp, "%f\t", snr);
			fprintf(fpd, "%f\t", snr);
			numb_snrs++;
			for (i = 0; i < size; i++)
			{
				decode_and_analyse(numb_of_bits, snr, iter_array[i], NO_OF_BLOCKS, guard_size, trellis_term_enable, decoder_type, guarding_type, &bit_error_count, &frame_error_count);
				fprintf(fp, "%d\t", bit_error_count);
				fprintf(gp, "%d\n", bit_error_count);
				fprintf(fpd, "%d\t", frame_error_count);
				fprintf(gpd, "%d\n", frame_error_count);
			}
			fprintf(fp, "\n");
			fprintf(fpd, "\n");
			printf("Done for snr = %f\n", snr);
		}
		fclose(fp);
		fclose(gp);
		fclose(fpd);
		fclose(gpd);
		// Converting the data into a format that can be used by gnuplot
		gp = fopen("ber_vs_snr_iters_linear.dat", "r");
		gpd = fopen("fer_vs_snr_iters_linear.dat", "r");
		bit_errordata = (int *)malloc(numb_snrs * size * sizeof(int));
		frame_errordata = (int *)malloc(numb_snrs * size * sizeof(int));
		for (i = 0; i < numb_snrs * size; i++)
		{
			fscanf(gp, "%d\n", &bit_errordata[i]);
			fscanf(gpd, "%d\n", &frame_errordata[i]);
		}
		fclose(gp);
		fclose(gpd);
		fp = fopen("ber_vs_snr_iters_gnu.dat", "w");
		fpd = fopen("fer_vs_snr_iters_gnu.dat", "w");
		for (k = 0; k < size; k++)
		{
			fprintf(fp, "#####Iteration %d#####\n", iter_array[k]);
			fprintf(fpd, "#####Iteration %d#####\n", iter_array[k]);
			for (i = 0; i < numb_snrs; i++)
			{
				fprintf(fp, "%f\t%d\t%g\n", snr_start + i * resolution, bit_errordata[k + i * size], bit_errordata[k + i * size] / (numb_of_bits * (1.0)));
				fprintf(fpd, "%f\t%d\t%g\n", snr_start + i * resolution, frame_errordata[k + i * size], frame_errordata[k + i * size] / (numb_of_frames * (1.0)));
			}
			fprintf(fp, "\n\n\n\n");
			fprintf(fpd, "\n\n\n\n");
		}
		fclose(fp);
		fclose(fpd);
		free(bit_errordata);
		free(frame_errordata);
	}
	ber_plotting_1(size, iter_array, decoder_type, guarding_type, guard_size);
	fer_plotting_1(size, iter_array, decoder_type, guarding_type, guard_size);
}

void simulation_blocks(int numb_of_bits, int decoder_type, int iter, int guarding_type, int guard_size, int trellis_term_enable, int only_plotting)
{
	void permuter_bits();
	void encoder_and_noise(int numb_of_bits, int trellis_termination, float snr, int noise_enable);
	void decode_and_analyse(int numb_of_bits, float snr, int iter_num, int blocks, int guard_size, int trellis_term_enable, int decoder_kind, int guarding_type, int *bit_error_count, int *frame_error_count);
	void ber_plotting_3(int size, int *blocks_array, int decoder_type, int iter, int guarding_type, int guard_size);
	void fer_plotting_3(int size, int *blocks_array, int decoder_type, int iter, int guarding_type, int guard_size);
	float snr;
	float snr_start = 0.0;
	float snr_end = 2.005;
	float resolution = 0.1;
	int blocks_array[5] = {32, 64, 96, 128, 192};
	int size = 5;
	int i, k, bit_error_count, frame_error_count, numb_snrs = 0, total_size;
	int *bit_errordata, *frame_errordata;
	FILE *fp, *gp, *fpd, *gpd;
	numb_of_bits = ((numb_of_bits + DATASIZE - 1) / DATASIZE) * DATASIZE;
	int numb_of_frames = numb_of_bits / DATASIZE;
	total_size = size;
	if (only_plotting != 1)
	{
		fp = fopen("ber_vs_snr_blocks.dat", "w");
		gp = fopen("ber_vs_snr_blocks_linear.dat", "w");
		fpd = fopen("fer_vs_snr_blocks.dat", "w");
		gpd = fopen("fer_vs_snr_blocks_linear.dat", "w");
		permuter_bits();
		for (snr = snr_start; snr <= snr_end; snr += resolution)
		{
			encoder_and_noise(numb_of_bits, 0, snr, 1);
			fprintf(fp, "%f\t", snr);
			fprintf(fpd, "%f\t", snr);
			numb_snrs++;
			for (i = 0; i < size; i++)
			{
				decode_and_analyse(numb_of_bits, snr, iter, blocks_array[i], guard_size, trellis_term_enable, decoder_type, guarding_type, &bit_error_count, &frame_error_count);
				fprintf(fp, "%d\t", bit_error_count);
				fprintf(gp, "%d\n", bit_error_count);
				fprintf(fpd, "%d\t", frame_error_count);
				fprintf(gpd, "%d\n", frame_error_count);
			}
			/*decode_and_analyse(numb_of_bits, snr, iter, 1, 0, trellis_term_enable, 1, 1, &bit_error_count, &frame_error_count); //on the cpu :(
			fprintf(fp, "%d\t", bit_error_count);
			fprintf(gp, "%d\n", bit_error_count);
			fprintf(fpd, "%d\t", frame_error_count);
			fprintf(gpd, "%d\n", frame_error_count);*/
			fprintf(fp, "\n");
			fprintf(fpd, "\n");
			printf("Done for snr = %f\n", snr);
		}
		fclose(fp);
		fclose(gp);
		fclose(fpd);
		fclose(gpd);
		// Converting the data into a format that can be used by gnuplot
		gp = fopen("ber_vs_snr_blocks_linear.dat", "r");
		gpd = fopen("fer_vs_snr_blocks_linear.dat", "r");
		bit_errordata = (int *)malloc(numb_snrs * total_size * sizeof(int));
		frame_errordata = (int *)malloc(numb_snrs * total_size * sizeof(int));
		for (i = 0; i < numb_snrs * total_size; i++)
		{
			fscanf(gp, "%d\n", &bit_errordata[i]);
			fscanf(gpd, "%d\n", &frame_errordata[i]);
		}
		fclose(gp);
		fclose(gpd);
		fp = fopen("ber_vs_snr_blocks_gnu.dat", "w");
		fpd = fopen("fer_vs_snr_blocks_gnu.dat", "w");
		for (k = 0; k < total_size; k++)
		{
			if (k == size)
			{
				fprintf(fp, "#####blocks 1#####\n");
				fprintf(fpd, "#####blocks 1#####\n");
			}
			else
			{
				fprintf(fp, "#####blocks %d#####\n", blocks_array[k]);
				fprintf(fpd, "#####blocks %d#####\n", blocks_array[k]);
			}
			for (i = 0; i < numb_snrs; i++)
			{
				fprintf(fp, "%f\t%d\t%g\n", snr_start + i * resolution, bit_errordata[k + i * total_size], bit_errordata[k + i * total_size] / (numb_of_bits * (1.0)));
				fprintf(fpd, "%f\t%d\t%g\n", snr_start + i * resolution, frame_errordata[k + i * total_size], frame_errordata[k + i * total_size] / (numb_of_frames * (1.0)));
			}
			fprintf(fp, "\n\n\n\n");
			fprintf(fpd, "\n\n\n\n");
		}
		fclose(fp);
		fclose(fpd);
		free(bit_errordata);
		free(frame_errordata);
	}
	ber_plotting_3(size, blocks_array, decoder_type, iter, guarding_type, guard_size);
	fer_plotting_3(size, blocks_array, decoder_type, iter, guarding_type, guard_size);
}

void simulation_guard(int numb_of_bits, int decoder_type, int iter, int trellis_term_enable, int only_plotting)
{
	// Plots BER vs SNR in logscale for different types of guarding and corresponding different guard sizes
	// Input information to be given is the number of iterations to be be performed
	void permuter_bits();
	void encoder_and_noise(int numb_of_bits, int trellis_termination, float snr, int noise_enable);
	void decode_and_analyse(int numb_of_bits, float snr, int iter_num, int blocks, int guard_size, int trellis_term_enable, int decoder_kind, int guarding_type, int *bit_error_count, int *frame_error_count);
	void ber_plotting_2(int size_2, int size_3, int *guard_size_2, int *guard_size_3, int decoder_type, int iter);
	void fer_plotting_2(int size_2, int size_3, int *guard_size_2, int *guard_size_3, int decoder_type, int iter);
	float snr;
	float snr_start = 0.0;
	float snr_end = 2.005;
	float resolution = 0.1;
	int i, k, bit_error_count, frame_error_count, numb_snrs = 0, total_size;
	int size_2 = 4;
	int size_3 = 3;
	int guard_size_2[4] = {3, 5, 8, 10};
	int guard_size_3[3] = {3, 5, 8};
	int *bit_errordata, *frame_errordata;
	FILE *fp, *gp, *fpd, *gpd;
	numb_of_bits = ((numb_of_bits + DATASIZE - 1) / DATASIZE) * DATASIZE;
	int numb_of_frames = numb_of_bits / DATASIZE;
	total_size = size_2 + size_3 + 2;
	if (only_plotting != 1)
	{
		fp = fopen("ber_vs_snr_guard.dat", "w");
		gp = fopen("ber_vs_snr_guard_linear.dat", "w");
		fpd = fopen("fer_vs_snr_guard.dat", "w");
		gpd = fopen("fer_vs_snr_guard_linear.dat", "w");
		permuter_bits();
		for (snr = snr_start; snr <= snr_end; snr += resolution)
		{
			encoder_and_noise(numb_of_bits, 0, snr, 1);
			fprintf(fp, "%f\t", snr);
			fprintf(fpd, "%f\t", snr);
			numb_snrs++;
			decode_and_analyse(numb_of_bits, snr, iter, NO_OF_BLOCKS, 0, trellis_term_enable, decoder_type, 1, &bit_error_count, &frame_error_count);
			fprintf(fp, "%d\t", bit_error_count);
			fprintf(gp, "%d\n", bit_error_count);
			fprintf(fpd, "%d\t", frame_error_count);
			fprintf(gpd, "%d\n", frame_error_count);
			for (i = 0; i < size_2; i++)
			{
				decode_and_analyse(numb_of_bits, snr, iter, NO_OF_BLOCKS, guard_size_2[i], trellis_term_enable, decoder_type, 2, &bit_error_count, &frame_error_count);
				fprintf(fp, "%d\t", bit_error_count);
				fprintf(gp, "%d\n", bit_error_count);
				fprintf(fpd, "%d\t", frame_error_count);
				fprintf(gpd, "%d\n", frame_error_count);
			}
			for (i = 0; i < size_3; i++)
			{
				decode_and_analyse(numb_of_bits, snr, iter, NO_OF_BLOCKS, guard_size_3[i], trellis_term_enable, decoder_type, 3, &bit_error_count, &frame_error_count);
				fprintf(fp, "%d\t", bit_error_count);
				fprintf(gp, "%d\n", bit_error_count);
				fprintf(fpd, "%d\t", frame_error_count);
				fprintf(gpd, "%d\n", frame_error_count);
			}

			decode_and_analyse(numb_of_bits, snr, iter, NO_OF_BLOCKS, 0, trellis_term_enable, decoder_type, 2, &bit_error_count, &frame_error_count);
			fprintf(fp, "%d\t", bit_error_count);
			fprintf(gp, "%d\n", bit_error_count);
			fprintf(fpd, "%d\t", frame_error_count);
			fprintf(gpd, "%d\n", frame_error_count);
			/*decode_and_analyse(numb_of_bits, snr, iter, NO_OF_BLOCKS, 0, trellis_term_enable, 1, 2, &bit_error_count, &frame_error_count); //Done on the cpu
			fprintf(fp, "%d\t", bit_error_count);
			fprintf(gp, "%d\n", bit_error_count);
			fprintf(fpd, "%d\t", frame_error_count);
			fprintf(gpd, "%d\n", frame_error_count);*/
			fprintf(fp, "\n");
			fprintf(fpd, "\n");
			printf("Done for snr = %f\n", snr);
		}
		fclose(fp);
		fclose(gp);
		fclose(fpd);
		fclose(gpd);
		// Converting the data into a format that can be used by gnuplot
		gp = fopen("ber_vs_snr_guard_linear.dat", "r");
		gpd = fopen("fer_vs_snr_guard_linear.dat", "r");
		bit_errordata = (int *)malloc(numb_snrs * total_size * sizeof(int));
		frame_errordata = (int *)malloc(numb_snrs * total_size * sizeof(int));
		for (i = 0; i < numb_snrs * total_size; i++)
		{
			fscanf(gp, "%d\n", &bit_errordata[i]);
			fscanf(gpd, "%d\n", &frame_errordata[i]);
		}
		fclose(gp);
		fclose(gpd);
		fp = fopen("ber_vs_snr_guard_gnu.dat", "w");
		fpd = fopen("fer_vs_snr_guard_gnu.dat", "w");
		for (k = 0; k < total_size; k++)
		{
			if (k == 0)
			{
				fprintf(fp, "#####Previous value initialisation#####\n");
				fprintf(fpd, "#####Previous value initialisation#####\n");
			}
			else if (k > 0 && k < 1 + size_2)
			{
				fprintf(fp, "#####Only training, guard size = %d#####\n", guard_size_2[k - 1]);
				fprintf(fpd, "#####Only training, guard size = %d#####\n", guard_size_2[k - 1]);
			}
			else if (k < total_size - 1)
			{
				fprintf(fp, "#####Previous value initialisation and training, guard size = %d#####\n", guard_size_2[k - 1 - size_2]);
				fprintf(fpd, "#####Previous value initialisation and training, guard size = %d#####\n", guard_size_2[k - 1 - size_2]);
			}
			else if (k == total_size - 1)
			{
				fprintf(fp, "####No guarding at all, equal value initialisation#####\n");
				fprintf(fpd, "####No guarding at all, equal value initialisation#####\n");
			}
			// else if(k == total_size - 1) fprintf(fp, "####Full max-log map decoding without parallelisation####\n");
			for (i = 0; i < numb_snrs; i++)
			{
				fprintf(fp, "%f\t%d\t%g\n", snr_start + i * resolution, bit_errordata[k + i * total_size], bit_errordata[k + i * total_size] / (numb_of_bits * (1.0)));
				fprintf(fpd, "%f\t%d\t%g\n", snr_start + i * resolution, frame_errordata[k + i * total_size], frame_errordata[k + i * total_size] / (numb_of_frames * (1.0)));
			}
			fprintf(fp, "\n\n\n\n");
			fprintf(fpd, "\n\n\n\n");
		}
		fclose(fp);
		fclose(fpd);
		free(bit_errordata);
		free(frame_errordata);
	}
	ber_plotting_2(size_2, size_3, guard_size_2, guard_size_3, decoder_type, iter);
	fer_plotting_2(size_2, size_3, guard_size_2, guard_size_3, decoder_type, iter);
}
