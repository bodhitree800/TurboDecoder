
#include <hip/hip_runtime.h>
#include <stdio.h>
// The following trellis functions are written specifically for
// g1(D)/g2(D) = (1 + D + D3 )/(1 + D2 + D3 )
// Need to be generalised
int next_state(int current_state, int input)
{
	int cpu_beta_state_0[8] = {0, 4, 5, 1, 2, 6, 7, 3};
	int cpu_beta_state_1[8] = {4, 0, 1, 5, 6, 2, 3, 7};
	int temp = 5;
	if (input == 0 || input == -1)
	{
		temp = cpu_beta_state_0[current_state];
	}
	else if (input == 1)
	{
		temp = cpu_beta_state_1[current_state];
	}
	return temp;
}

int output(int current_state, int input)
{
	int cpu_beta_encbit_0[8] = {-1, -1, 1, 1, 1, 1, -1, -1};
	int cpu_beta_encbit_1[8] = {1, 1, -1, -1, -1, -1, 1, 1};
	int temp = 5;
	if (input == 0 || input == -1)
	{
		temp = cpu_beta_encbit_0[current_state];
	}
	else if (input == 1)
	{
		temp = cpu_beta_encbit_1[current_state];
	}
	return temp;
}
